#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 15 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"




/* Line 189 of yacc.c  */
#line 84 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     NONEQUAL = 268,
     OR = 269,
     XOR = 270,
     AND = 271,
     DISTINCT = 272,
     YEAR = 273,
     MONTH = 274,
     DAY = 275,
     REGEXP = 276,
     LIKE = 277,
     IS = 278,
     IN = 279,
     NOT = 280,
     BETWEEN = 281,
     COMPARISON = 282,
     SHIFT = 283,
     MOD = 284,
     FROM = 285,
     DELETE = 286,
     LOAD = 287,
     FILTER = 288,
     BY = 289,
     JOIN = 290,
     STORE = 291,
     INTO = 292,
     GROUP = 293,
     SELECT = 294,
     AS = 295,
     ORDER = 296,
     ASC = 297,
     DESC = 298,
     COUNT = 299,
     USING = 300,
     SUM = 301,
     AVG = 302,
     MIN = 303,
     MAX = 304,
     LIMIT = 305,
     ON = 306,
     BINARY = 307,
     LEFT = 308,
     RIGHT = 309,
     OUTER = 310,
     SEMI = 311,
     ANTI = 312,
     SORT = 313,
     SEGMENTS = 314,
     PRESORTED = 315,
     PARTITION = 316,
     INSERT = 317,
     WHERE = 318,
     DISPLAY = 319,
     CASE = 320,
     WHEN = 321,
     THEN = 322,
     ELSE = 323,
     END = 324,
     SHOW = 325,
     TABLES = 326,
     TABLE = 327,
     DESCRIBE = 328,
     DROP = 329,
     CREATE = 330,
     INDEX = 331,
     INTERVAL = 332,
     APPEND = 333
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 25 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 207 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 219 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  23
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   926

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  96
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  99
/* YYNRULES -- Number of states.  */
#define YYNSTATES  297

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   333

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    25,     2,     2,     2,    36,    30,     2,
      89,    90,    34,    32,    92,    33,    91,    35,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    95,    88,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    38,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    93,    29,    94,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    21,    22,    23,    24,
      26,    27,    28,    31,    37,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81,    82,    83,
      84,    85,    86,    87
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    82,    85,    93,    99,   107,   110,   114,
     137,   146,   157,   159,   163,   165,   167,   169,   171,   173,
     175,   187,   197,   204,   207,   210,   215,   220,   225,   230,
     235,   238,   243,   248,   253,   258,   262,   266,   270,   274,
     278,   282,   286,   290,   294,   298,   302,   306,   309,   312,
     316,   320,   326,   330,   339,   343,   348,   349,   353,   357,
     363,   365,   367,   371,   373,   377,   378,   380,   383,   388,
     395,   402,   409,   415,   421,   428,   434,   440,   448,   456,
     463,   471,   478,   486,   493,   494,   497,   498,   503,   511
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      97,     0,    -1,    98,    88,    -1,    97,    98,    88,    -1,
      99,    -1,     4,    11,    48,   102,    39,     4,   101,    -1,
       4,    11,    41,     3,    54,    89,     3,    90,    49,    89,
     103,    90,    -1,     4,    11,    42,     4,   106,    -1,     4,
      11,    50,     4,    43,   105,    -1,     4,    11,    48,   102,
      39,     4,   107,   101,    -1,    45,     4,    46,     3,    54,
      89,     3,    90,   108,    -1,    45,     4,    46,     3,   108,
      61,   109,    -1,    45,     4,    46,     3,    87,   108,    61,
     109,    -1,    82,     4,    -1,    71,    46,     4,    48,   102,
      39,     4,    -1,    40,    39,     4,    72,   100,    -1,    73,
       4,    54,    89,     3,    90,   108,    -1,    79,    80,    -1,
      83,    81,     4,    -1,    84,    85,     4,    60,     4,    89,
       4,    91,     4,    90,    39,     4,    92,     4,    72,     4,
      91,     4,    12,     4,    91,     4,    -1,    84,    85,     4,
      60,     4,    89,     4,    90,    -1,    84,    86,     4,    60,
       4,    89,     4,    92,     4,    90,    -1,     4,    -1,     4,
      91,     4,    -1,    10,    -1,     5,    -1,     6,    -1,     7,
      -1,     9,    -1,     8,    -1,     4,    93,     6,    94,    95,
       4,    89,     6,    92,     6,    90,    -1,     4,    93,     6,
      94,    95,     4,    89,     6,    90,    -1,     4,    93,     6,
      94,    95,     4,    -1,     4,    51,    -1,     4,    52,    -1,
      53,    89,   100,    90,    -1,    55,    89,   100,    90,    -1,
      56,    89,   100,    90,    -1,    57,    89,   100,    90,    -1,
      58,    89,   100,    90,    -1,    17,   100,    -1,    18,    89,
     100,    90,    -1,    19,    89,   100,    90,    -1,    20,    89,
     100,    90,    -1,     4,    89,     5,    90,    -1,   100,    32,
     100,    -1,   100,    33,   100,    -1,   100,    34,   100,    -1,
     100,    35,   100,    -1,   100,    36,   100,    -1,   100,    37,
     100,    -1,   100,    16,   100,    -1,   100,    12,   100,    -1,
     100,    13,   100,    -1,   100,    14,   100,    -1,   100,    15,
     100,    -1,   100,    31,   100,    -1,    26,   100,    -1,    25,
     100,    -1,   100,    28,   100,    -1,   100,    22,   100,    -1,
     100,    28,    89,    99,    90,    -1,    89,   100,    90,    -1,
      74,    75,   100,    76,   100,    77,   100,    78,    -1,   100,
      23,     8,    -1,   100,    23,    26,     8,    -1,    -1,    47,
      43,   104,    -1,   100,    49,     4,    -1,   102,    92,   100,
      49,     4,    -1,    34,    -1,   100,    -1,   103,    92,   100,
      -1,   100,    -1,   100,    92,   104,    -1,    -1,   104,    -1,
      43,   100,    -1,    44,     4,    60,   100,    -1,    62,    66,
      44,     4,    60,   100,    -1,    63,    66,    44,     4,    60,
     100,    -1,    62,    65,    44,     4,    60,   100,    -1,    62,
      44,     4,    60,   100,    -1,    63,    44,     4,    60,   100,
      -1,    63,    65,    44,     4,    60,   100,    -1,    64,    44,
       4,    60,   100,    -1,    44,     4,    60,   100,   107,    -1,
      62,    66,    44,     4,    60,   100,   107,    -1,    63,    66,
      44,     4,    60,   100,   107,    -1,    62,    44,     4,    60,
     100,   107,    -1,    62,    65,    44,     4,    60,   100,   107,
      -1,    63,    44,     4,    60,   100,   107,    -1,    63,    65,
      44,     4,    60,   100,   107,    -1,    64,    44,     4,    60,
     100,   107,    -1,    -1,    59,     6,    -1,    -1,    67,    68,
      43,     4,    -1,    67,    68,    43,     4,    70,    43,     6,
      -1,    69,    43,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   128,   128,   129,   133,   136,   138,   140,   142,   144,
     146,   148,   150,   152,   154,   156,   158,   160,   162,   164,
     166,   168,   174,   175,   176,   177,   178,   179,   180,   181,
     182,   183,   184,   185,   186,   187,   188,   189,   190,   191,
     192,   193,   194,   195,   196,   200,   201,   202,   203,   204,
     205,   206,   207,   208,   209,   210,   211,   212,   213,   214,
     215,   217,   218,   219,   223,   224,   227,   230,   234,   235,
     236,   240,   241,   245,   246,   249,   251,   254,   258,   259,
     260,   261,   262,   263,   264,   265,   266,   267,   268,   269,
     270,   271,   272,   273,   275,   278,   280,   283,   284,   285
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL",
  "NONEQUAL", "OR", "XOR", "AND", "DISTINCT", "YEAR", "MONTH", "DAY",
  "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT", "BETWEEN", "COMPARISON",
  "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'", "'/'", "'%'", "MOD", "'^'",
  "FROM", "DELETE", "LOAD", "FILTER", "BY", "JOIN", "STORE", "INTO",
  "GROUP", "SELECT", "AS", "ORDER", "ASC", "DESC", "COUNT", "USING", "SUM",
  "AVG", "MIN", "MAX", "LIMIT", "ON", "BINARY", "LEFT", "RIGHT", "OUTER",
  "SEMI", "ANTI", "SORT", "SEGMENTS", "PRESORTED", "PARTITION", "INSERT",
  "WHERE", "DISPLAY", "CASE", "WHEN", "THEN", "ELSE", "END", "SHOW",
  "TABLES", "TABLE", "DESCRIBE", "DROP", "CREATE", "INDEX", "INTERVAL",
  "APPEND", "';'", "'('", "')'", "'.'", "','", "'{'", "'}'", "':'",
  "$accept", "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,   276,   277,   278,   279,    33,   280,   281,   282,   124,
      38,   283,    43,    45,    42,    47,    37,   284,    94,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,   328,   329,   330,   331,   332,   333,    59,    40,
      41,    46,    44,   123,   125,    58
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    96,    97,    97,    98,    99,    99,    99,    99,    99,
      99,    99,    99,    99,    99,    99,    99,    99,    99,    99,
      99,    99,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   101,   101,   102,   102,
     102,   103,   103,   104,   104,   105,   105,   106,   107,   107,
     107,   107,   107,   107,   107,   107,   107,   107,   107,   107,
     107,   107,   107,   107,   108,   108,   109,   109,   109,   109
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     8,     2,     7,     5,     7,     2,     3,    22,
       8,    10,     1,     3,     1,     1,     1,     1,     1,     1,
      11,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       2,     4,     4,     4,     4,     3,     3,     3,     3,     3,
       3,     3,     3,     3,     3,     3,     3,     2,     2,     3,
       3,     5,     3,     8,     3,     4,     0,     3,     3,     5,
       1,     1,     3,     1,     3,     0,     1,     2,     4,     6,
       6,     6,     5,     5,     6,     5,     5,     7,     7,     6,
       7,     6,     7,     6,     0,     2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    17,    13,
       0,     0,     0,     1,     0,     2,     0,     0,     0,     0,
       0,     0,     0,     0,    18,     0,     0,     3,     0,     0,
      22,    25,    26,    27,    29,    28,    24,     0,     0,     0,
       0,     0,     0,    70,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,    94,     0,     0,     0,     0,
       0,     0,     7,    33,    34,     0,     0,     0,    40,     0,
       0,     0,    58,    57,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,    75,
      15,     0,     0,    94,     0,     0,     0,     0,     0,     0,
      77,     0,    23,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,    62,    52,    53,    54,    55,    51,    60,
      64,     0,     0,    59,    56,    45,    46,    47,    48,    49,
      50,    68,    66,     0,    73,    76,     8,     0,    95,     0,
      96,     0,    94,     0,     0,     0,    44,     0,    41,    42,
      43,    35,    36,    37,    38,    39,     0,    65,    22,     0,
       0,     0,     0,     0,     0,     5,    66,     0,     0,     0,
      96,     0,     0,    11,    14,    16,     0,     0,     0,     0,
       0,    61,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     9,    69,    74,    94,    12,     0,     0,    20,     0,
       0,     0,    32,     0,     0,    67,     0,     0,     0,     0,
       0,     0,     0,    10,     0,    99,     0,     0,     0,     0,
       0,    78,     0,     0,     0,     0,     0,     0,     0,    97,
       0,    21,    71,     0,     0,    63,    86,    82,     0,     0,
      83,     0,     0,    85,     0,     0,     6,     0,    31,     0,
      89,    81,    79,    91,    84,    80,    93,     0,     0,    72,
       0,    90,    87,    92,    88,    98,     0,    30,     0,     0,
       0,     0,     0,     0,     0,     0,    19
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   154,   185,    62,   253,   155,   156,
      72,   186,   114,   193
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -182
static const yytype_int16 yypact[] =
{
     118,     9,   -29,    27,     8,    41,   -11,    44,    -3,    14,
     264,     1,  -182,    -1,    87,    61,   107,    58,  -182,  -182,
     109,   111,   112,  -182,    29,  -182,   116,   117,   120,   128,
      48,   140,    96,    64,  -182,    95,    99,  -182,    93,   113,
     -47,  -182,  -182,  -182,  -182,  -182,  -182,   218,    75,    76,
      77,   218,   218,  -182,    79,    80,    81,    82,    83,   105,
     218,   743,   -37,   131,   218,   -48,   120,   147,   178,   180,
      97,   218,  -182,  -182,  -182,   182,   181,   184,   863,   218,
     218,   218,   889,   889,   218,   218,   218,   218,   218,   218,
     363,   218,   218,   218,   218,   218,   218,    -5,   241,   218,
     218,   218,   218,   218,   218,   218,   188,   189,   218,   218,
     799,   110,   192,   129,   142,   -30,   114,   119,   122,   202,
     799,   123,  -182,   121,   389,   423,   449,   475,   509,   535,
     561,   595,   674,  -182,   799,   824,   847,   290,   863,   879,
    -182,   198,    78,   246,   -19,    -9,    -9,  -182,  -182,  -182,
    -182,  -182,    46,   771,   337,  -182,  -182,   204,  -182,   151,
      12,   212,   129,   214,   215,   139,  -182,   126,  -182,  -182,
    -182,  -182,  -182,  -182,  -182,  -182,   218,  -182,    90,   150,
     228,   199,   -36,   -32,   197,  -182,   205,   249,   218,   164,
      12,   187,   213,  -182,  -182,  -182,    15,   165,   216,   258,
     647,  -182,   203,   218,   265,   226,   240,   268,   243,   244,
     281,  -182,  -182,  -182,   129,  -182,   247,   285,  -182,   287,
     289,   206,   211,   218,   218,  -182,   250,   297,   299,   251,
     310,   324,   269,  -182,   327,  -182,   242,   248,   218,   328,
     621,   700,   218,   273,   276,   218,   279,   280,   218,   271,
     303,  -182,   799,   -55,     2,  -182,  -182,   700,   218,   218,
     700,   218,   218,   700,   301,   341,  -182,   218,  -182,   348,
    -182,   700,   700,  -182,   700,   700,  -182,   349,   266,   799,
     267,  -182,  -182,  -182,  -182,  -182,   352,  -182,   291,   357,
     275,   358,   355,   360,   292,   376,  -182
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -182,  -182,   371,   245,   -28,   196,   318,  -182,  -181,  -182,
    -182,    45,  -112,   200
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      61,   159,   107,   140,    73,    74,   111,   213,   204,   161,
      14,   112,   207,   100,   101,   102,   103,   104,   105,    78,
      13,   141,   225,    82,    83,   102,   103,   104,   105,   205,
     206,    15,    90,   208,   209,   266,   110,   267,    61,   113,
      26,    27,    75,   120,    76,    17,    77,    28,    19,    29,
     195,   124,   125,   126,    16,   108,   127,   128,   129,   130,
     131,   132,   108,   134,   135,   136,   137,   138,   139,    18,
     143,   144,   145,   146,   147,   148,   149,   150,    20,   191,
     153,   192,   178,    41,    42,    43,    44,    45,    46,    25,
     180,    30,   268,   181,   269,    47,    48,    49,    50,    21,
      22,    13,   233,    51,    52,   218,   219,    31,   182,   183,
     184,    32,    33,    34,    90,    35,    36,    37,     2,    38,
      64,    39,     1,     3,    40,    41,    42,    43,    44,    45,
      46,    54,    63,    55,    56,    57,    58,    47,    48,    49,
      50,    73,    74,    65,    66,    51,    52,    70,   200,     4,
     116,     5,    59,    67,    53,    68,    71,     6,     2,    69,
       7,     8,     9,     3,    79,    80,    81,    60,    84,    85,
      86,    87,    88,    54,   109,    55,    56,    57,    58,    75,
      89,    76,   117,    77,   118,   122,   119,   121,   112,     4,
     123,     5,   151,   152,    59,   240,   241,     6,   158,   157,
       7,     8,     9,   160,   162,   165,   177,   189,   163,    60,
     252,   164,   190,   166,   257,   167,   194,   260,   196,   197,
     263,   199,    40,    41,    42,    43,    44,    45,    46,   198,
     271,   272,   202,   274,   275,    47,    48,    49,    50,   279,
     201,   210,   203,    51,    52,    40,    41,    42,    43,    44,
      45,    46,   181,   212,   214,   216,   217,   220,    47,    48,
      49,    50,   222,   224,    23,   221,    51,    52,     1,   226,
     227,    54,   229,    55,    56,    57,    58,    99,   100,   101,
     102,   103,   104,   105,   228,   232,   256,   230,   231,   235,
     234,   236,    59,   237,    54,   238,    55,    56,    57,    58,
     239,   243,   270,   244,     2,   273,    95,    60,   276,     3,
     242,   245,    96,    97,   246,    59,   281,   282,    98,   283,
     284,    99,   100,   101,   102,   103,   104,   105,   247,   248,
     142,   249,   250,   258,   254,     4,   259,     5,   251,   261,
     262,   264,   265,     6,   277,   278,     7,     8,     9,    91,
      92,    93,    94,    95,   280,   285,   288,   287,   286,    96,
      97,   290,   292,   289,   294,    98,   291,   293,    99,   100,
     101,   102,   103,   104,   105,    91,    92,    93,    94,    95,
     296,    24,   211,   295,   115,    96,    97,   179,     0,     0,
     215,    98,     0,     0,    99,   100,   101,   102,   103,   104,
     105,    91,    92,    93,    94,    95,     0,     0,     0,     0,
       0,    96,    97,     0,     0,     0,     0,    98,     0,     0,
      99,   100,   101,   102,   103,   104,   105,     0,     0,   188,
       0,     0,     0,     0,     0,    91,    92,    93,    94,    95,
       0,     0,     0,     0,     0,    96,    97,     0,     0,     0,
       0,    98,     0,   133,    99,   100,   101,   102,   103,   104,
     105,    91,    92,    93,    94,    95,     0,     0,     0,     0,
       0,    96,    97,     0,     0,     0,     0,    98,     0,   168,
      99,   100,   101,   102,   103,   104,   105,    91,    92,    93,
      94,    95,     0,     0,     0,     0,     0,    96,    97,     0,
       0,     0,     0,    98,     0,     0,    99,   100,   101,   102,
     103,   104,   105,   169,     0,     0,     0,     0,     0,     0,
       0,    91,    92,    93,    94,    95,     0,     0,     0,     0,
       0,    96,    97,     0,     0,     0,     0,    98,     0,   170,
      99,   100,   101,   102,   103,   104,   105,    91,    92,    93,
      94,    95,     0,     0,     0,     0,     0,    96,    97,     0,
       0,     0,     0,    98,     0,   171,    99,   100,   101,   102,
     103,   104,   105,    91,    92,    93,    94,    95,     0,     0,
       0,     0,     0,    96,    97,     0,     0,     0,     0,    98,
       0,     0,    99,   100,   101,   102,   103,   104,   105,   172,
       0,     0,     0,     0,     0,     0,     0,    91,    92,    93,
      94,    95,     0,     0,     0,     0,     0,    96,    97,     0,
       0,     0,     0,    98,     0,   173,    99,   100,   101,   102,
     103,   104,   105,    91,    92,    93,    94,    95,     0,     0,
       0,     0,     0,    96,    97,     0,     0,     0,     0,    98,
       0,   174,    99,   100,   101,   102,   103,   104,   105,    91,
      92,    93,    94,    95,     0,     0,     0,     0,     0,    96,
      97,     0,     0,     0,     0,    98,     0,     0,    99,   100,
     101,   102,   103,   104,   105,   175,    91,    92,    93,    94,
      95,     0,     0,     0,     0,     0,    96,    97,     0,   255,
       0,     0,    98,     0,     0,    99,   100,   101,   102,   103,
     104,   105,    91,    92,    93,    94,    95,     0,     0,     0,
       0,     0,    96,    97,   223,     0,     0,     0,    98,     0,
       0,    99,   100,   101,   102,   103,   104,   105,     0,     0,
       0,     0,     0,     0,   180,     0,     0,     0,     0,     0,
     176,     0,     0,     0,     0,    91,    92,    93,    94,    95,
       0,     0,   182,   183,   184,    96,    97,     0,     0,     0,
       0,    98,     0,     0,    99,   100,   101,   102,   103,   104,
     105,     0,     0,    91,    92,    93,    94,    95,     0,     0,
       0,     0,   106,    96,    97,     0,     0,     0,     0,    98,
       0,     0,    99,   100,   101,   102,   103,   104,   105,     0,
       0,    91,    92,    93,    94,    95,     0,     0,     0,     0,
     187,    96,    97,     0,     0,     0,     0,    98,     0,     0,
      99,   100,   101,   102,   103,   104,   105,    92,    93,    94,
      95,     0,     0,     0,     0,     0,    96,    97,     0,     0,
       0,     0,    98,     0,     0,    99,   100,   101,   102,   103,
     104,   105,    94,    95,     0,     0,     0,     0,     0,    96,
      97,     0,     0,     0,     0,    98,     0,     0,    99,   100,
     101,   102,   103,   104,   105,    96,    97,     0,     0,     0,
       0,    98,     0,     0,    99,   100,   101,   102,   103,   104,
     105,    -1,    -1,     0,     0,     0,     0,    98,     0,     0,
      99,   100,   101,   102,   103,   104,   105,    98,     0,     0,
      99,   100,   101,   102,   103,   104,   105
};

static const yytype_int16 yycheck[] =
{
      28,   113,    39,     8,    51,    52,    54,   188,    44,    39,
      39,    59,    44,    32,    33,    34,    35,    36,    37,    47,
      11,    26,   203,    51,    52,    34,    35,    36,    37,    65,
      66,     4,    60,    65,    66,    90,    64,    92,    66,    87,
      41,    42,    89,    71,    91,     4,    93,    48,     4,    50,
     162,    79,    80,    81,    46,    92,    84,    85,    86,    87,
      88,    89,    92,    91,    92,    93,    94,    95,    96,    80,
      98,    99,   100,   101,   102,   103,   104,   105,    81,    67,
     108,    69,     4,     5,     6,     7,     8,     9,    10,    88,
      44,     4,    90,    47,    92,    17,    18,    19,    20,    85,
      86,    11,   214,    25,    26,    90,    91,    46,    62,    63,
      64,     4,    54,     4,   142,     4,     4,    88,    40,     3,
      72,     4,     4,    45,     4,     5,     6,     7,     8,     9,
      10,    53,     4,    55,    56,    57,    58,    17,    18,    19,
      20,    51,    52,     3,    48,    25,    26,    54,   176,    71,
       3,    73,    74,    89,    34,    60,    43,    79,    40,    60,
      82,    83,    84,    45,    89,    89,    89,    89,    89,    89,
      89,    89,    89,    53,    43,    55,    56,    57,    58,    89,
      75,    91,     4,    93,     4,     4,    89,     5,    59,    71,
       6,    73,     4,     4,    74,   223,   224,    79,     6,    89,
      82,    83,    84,    61,    90,     3,     8,     3,    89,    89,
     238,    89,    61,    90,   242,    94,     4,   245,     4,     4,
     248,    95,     4,     5,     6,     7,     8,     9,    10,    90,
     258,   259,     4,   261,   262,    17,    18,    19,    20,   267,
      90,    44,    43,    25,    26,     4,     5,     6,     7,     8,
       9,    10,    47,     4,    90,    68,    43,    92,    17,    18,
      19,    20,     4,    60,     0,    49,    25,    26,     4,     4,
      44,    53,     4,    55,    56,    57,    58,    31,    32,    33,
      34,    35,    36,    37,    44,     4,   241,    44,    44,     4,
      43,     4,    74,     4,    53,    89,    55,    56,    57,    58,
      89,     4,   257,     4,    40,   260,    16,    89,   263,    45,
      60,    60,    22,    23,     4,    74,   271,   272,    28,   274,
     275,    31,    32,    33,    34,    35,    36,    37,     4,    60,
      89,     4,    90,    60,     6,    71,    60,    73,    90,    60,
      60,    70,    39,    79,    43,     4,    82,    83,    84,    12,
      13,    14,    15,    16,     6,     6,     4,    90,    92,    22,
      23,     4,     4,    72,     4,    28,    91,    12,    31,    32,
      33,    34,    35,    36,    37,    12,    13,    14,    15,    16,
       4,    10,   186,    91,    66,    22,    23,   142,    -1,    -1,
     190,    28,    -1,    -1,    31,    32,    33,    34,    35,    36,
      37,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    -1,    -1,    92,
      -1,    -1,    -1,    -1,    -1,    12,    13,    14,    15,    16,
      -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    90,    31,    32,    33,    34,    35,    36,
      37,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    90,
      31,    32,    33,    34,    35,    36,    37,    12,    13,    14,
      15,    16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,
      -1,    -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,
      35,    36,    37,    90,    -1,    -1,    -1,    -1,    -1,    -1,
      -1,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    90,
      31,    32,    33,    34,    35,    36,    37,    12,    13,    14,
      15,    16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,
      -1,    -1,    -1,    28,    -1,    90,    31,    32,    33,    34,
      35,    36,    37,    12,    13,    14,    15,    16,    -1,    -1,
      -1,    -1,    -1,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    -1,    31,    32,    33,    34,    35,    36,    37,    90,
      -1,    -1,    -1,    -1,    -1,    -1,    -1,    12,    13,    14,
      15,    16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,
      -1,    -1,    -1,    28,    -1,    90,    31,    32,    33,    34,
      35,    36,    37,    12,    13,    14,    15,    16,    -1,    -1,
      -1,    -1,    -1,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    90,    31,    32,    33,    34,    35,    36,    37,    12,
      13,    14,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    -1,    31,    32,
      33,    34,    35,    36,    37,    90,    12,    13,    14,    15,
      16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    78,
      -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,    35,
      36,    37,    12,    13,    14,    15,    16,    -1,    -1,    -1,
      -1,    -1,    22,    23,    77,    -1,    -1,    -1,    28,    -1,
      -1,    31,    32,    33,    34,    35,    36,    37,    -1,    -1,
      -1,    -1,    -1,    -1,    44,    -1,    -1,    -1,    -1,    -1,
      76,    -1,    -1,    -1,    -1,    12,    13,    14,    15,    16,
      -1,    -1,    62,    63,    64,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    -1,    31,    32,    33,    34,    35,    36,
      37,    -1,    -1,    12,    13,    14,    15,    16,    -1,    -1,
      -1,    -1,    49,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    -1,    31,    32,    33,    34,    35,    36,    37,    -1,
      -1,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      49,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    13,    14,    15,
      16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,
      -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,    35,
      36,    37,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    -1,    31,    32,
      33,    34,    35,    36,    37,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    -1,    31,    32,    33,    34,    35,    36,
      37,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    40,    45,    71,    73,    79,    82,    83,    84,
      97,    98,    99,    11,    39,     4,    46,     4,    80,     4,
      81,    85,    86,     0,    98,    88,    41,    42,    48,    50,
       4,    46,     4,    54,     4,     4,     4,    88,     3,     4,
       4,     5,     6,     7,     8,     9,    10,    17,    18,    19,
      20,    25,    26,    34,    53,    55,    56,    57,    58,    74,
      89,   100,   102,     4,    72,     3,    48,    89,    60,    60,
      54,    43,   106,    51,    52,    89,    91,    93,   100,    89,
      89,    89,   100,   100,    89,    89,    89,    89,    89,    75,
     100,    12,    13,    14,    15,    16,    22,    23,    28,    31,
      32,    33,    34,    35,    36,    37,    49,    39,    92,    43,
     100,    54,    59,    87,   108,   102,     3,     4,     4,    89,
     100,     5,     4,     6,   100,   100,   100,   100,   100,   100,
     100,   100,   100,    90,   100,   100,   100,   100,   100,   100,
       8,    26,    89,   100,   100,   100,   100,   100,   100,   100,
     100,     4,     4,   100,   100,   104,   105,    89,     6,   108,
      61,    39,    90,    89,    89,     3,    90,    94,    90,    90,
      90,    90,    90,    90,    90,    90,    76,     8,     4,    99,
      44,    47,    62,    63,    64,   101,   107,    49,    92,     3,
      61,    67,    69,   109,     4,   108,     4,     4,    90,    95,
     100,    90,     4,    43,    44,    65,    66,    44,    65,    66,
      44,   101,     4,   104,    90,   109,    68,    43,    90,    91,
      92,    49,     4,    77,    60,   104,     4,    44,    44,     4,
      44,    44,     4,   108,    43,     4,     4,     4,    89,    89,
     100,   100,    60,     4,     4,    60,     4,     4,    60,     4,
      90,    90,   100,   103,     6,    78,   107,   100,    60,    60,
     100,    60,    60,   100,    70,    39,    90,    92,    90,    92,
     107,   100,   100,   107,   100,   100,   107,    43,     4,   100,
       6,   107,   107,   107,   107,     6,    92,    90,     4,    72,
       4,    91,     4,    12,     4,    91,     4
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 133 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 137 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 139 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 141 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 143 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval),0,-1); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 147 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval),0); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (8)].strval),(yyvsp[(4) - (8)].strval),1); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 157 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    {  emit_show_tables();;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    {  emit_create_bitmap_index((yyvsp[(3) - (22)].strval), (yyvsp[(5) - (22)].strval), (yyvsp[(7) - (22)].strval), (yyvsp[(9) - (22)].strval), (yyvsp[(18) - (22)].strval), (yyvsp[(22) - (22)].strval));;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    {  emit_create_index((yyvsp[(3) - (8)].strval), (yyvsp[(5) - (8)].strval), (yyvsp[(7) - (8)].strval));;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    {  emit_create_interval((yyvsp[(3) - (10)].strval), (yyvsp[(5) - (10)].strval), (yyvsp[(7) - (10)].strval), (yyvsp[(9) - (10)].strval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_fieldname((yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 181 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_vardecimal((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval),  (yyvsp[(8) - (11)].intval), (yyvsp[(10) - (11)].intval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit_count(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit_sum(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_average(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_min(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_max(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit_distinct(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit_year(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit_month(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit_day(); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_string_grp((yyvsp[(1) - (4)].strval), (yyvsp[(3) - (4)].strval)); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    { emit_add(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_minus(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    { emit_mul(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 203 "bison.y"
    { emit_div(); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    { emit("MOD"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("MOD"); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit_and(); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 207 "bison.y"
    { emit_eq(); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 208 "bison.y"
    { emit_neq(); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { emit_or(); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("XOR"); ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { emit("NOT"); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 213 "bison.y"
    { emit("NOT"); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 214 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 215 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    {emit("EXPR");;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 219 "bison.y"
    { emit_case(); ;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 224 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 230 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 234 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 69:

/* Line 1455 of yacc.c  */
#line 235 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 241 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 254 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '3');;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 260 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '4');;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 261 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '1');;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 262 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'S');;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 263 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 264 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '2');;}
    break;

  case 85:

/* Line 1455 of yacc.c  */
#line 265 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 86:

/* Line 1455 of yacc.c  */
#line 266 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 87:

/* Line 1455 of yacc.c  */
#line 267 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '3'); ;}
    break;

  case 88:

/* Line 1455 of yacc.c  */
#line 268 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '4'); ;}
    break;

  case 89:

/* Line 1455 of yacc.c  */
#line 269 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 90:

/* Line 1455 of yacc.c  */
#line 270 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '1'); ;}
    break;

  case 91:

/* Line 1455 of yacc.c  */
#line 271 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 92:

/* Line 1455 of yacc.c  */
#line 272 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), 'R'); ;}
    break;

  case 93:

/* Line 1455 of yacc.c  */
#line 273 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 94:

/* Line 1455 of yacc.c  */
#line 275 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 95:

/* Line 1455 of yacc.c  */
#line 278 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 96:

/* Line 1455 of yacc.c  */
#line 280 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 97:

/* Line 1455 of yacc.c  */
#line 283 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 98:

/* Line 1455 of yacc.c  */
#line 284 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 99:

/* Line 1455 of yacc.c  */
#line 285 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2459 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 287 "bison.y"


bool scan_state;
unsigned int statement_count;
time_t curr_time;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;
    process_count = 1000000000; //1GB by default
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = 1000000*atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };

    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();
		filter_var.clear();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, nullptr, verbose);

		curr_time = time(0)*1000;
        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
			filter_var.clear();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();
			curr_time = time(0)*1000;
			
            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
			hipHostFree(buffers[buffer_names.front()]);
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };
		for(auto it = idx_vals.begin(); it != idx_vals.end();it++) {
			hipFree(it->second);
		idx_vals.clear();	
    };
	

    };
    if(save_dict) {
        save_col_data(data_dict,"data.dictionary");
	};	

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
	if(scratch.size()) {
		scratch.resize(0);
		scratch.shrink_to_fit();
	};	
	if(rcol_dev.size()) {
		rcol_dev.resize(0);
		rcol_dev.shrink_to_fit();
	};
	if(ranj.size()) {
		ranj.resize(0);
		ranj.shrink_to_fit();
	};	
    return 0;
}



//external c global to report errors
//char alenka_err[4048];


int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}



