#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cm.h"
#include "zone_map.h"


using namespace mgpu;

vector<void*> alloced_mem;

template<typename T>
struct distinct : public binary_function<T,T,T>
{
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return lhs != rhs;
    }
};


struct gpu_getyear
{
	const int_type *source;
    int_type *dest;
		
	gpu_getyear(const int_type *_source, int_type *_dest):
			  source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {	
		
		uint64 sec;
		uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
		uint year, leap;
		uint yday;
		uint month, mday;
		const uint daysSinceJan1st[2][13]=
		{
			{0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
			{0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
		};
		uint64 SecondsSinceEpoch = source[i]/1000;
		sec = SecondsSinceEpoch + 11644473600;

		//wday = (uint)((sec / 86400 + 1) % 7); // day of week
		quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
		sec %= 12622780800ULL;

		centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
		if (centennials > 3)
		{
			centennials = 3;
		}
		sec -= centennials * 3155673600ULL;

		quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
		if (quadrennials > 24)
		{
			quadrennials = 24;
		}
		sec -= quadrennials * 126230400ULL;

		annuals = (uint)(sec / 31536000); // 365*24*3600
		if (annuals > 3)
		{
			annuals = 3;
		}
		sec -= annuals * 31536000ULL;

		year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
		leap = !(year % 4) && (year % 100 || !(year % 400));

		// Calculate the day of the year and the time
		yday = sec / 86400;
		sec %= 86400;
		//hour = sec / 3600;
		sec %= 3600;
		//min = sec / 60;
		sec %= 60;

	// Calculate the month
		for (mday = month = 1; month < 13; month++)
		{
			if (yday < daysSinceJan1st[leap][month])
			{
			mday += yday - daysSinceJan1st[leap][month - 1];
			break;
			}
		}
		dest[i] = year;		
	}
};	

struct gpu_getmonth
{
	const int_type *source;
    int_type *dest;
		
	gpu_getmonth(const int_type *_source, int_type *_dest):
			  source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {	
		
		uint64 sec;
		uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
		uint year, leap;
		uint yday;
		uint month, mday;
		const uint daysSinceJan1st[2][13]=
		{
			{0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
			{0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
		};
		uint64 SecondsSinceEpoch = source[i]/1000;
		sec = SecondsSinceEpoch + 11644473600;

		//wday = (uint)((sec / 86400 + 1) % 7); // day of week
		quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
		sec %= 12622780800ULL;

		centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
		if (centennials > 3)
		{
			centennials = 3;
		}
		sec -= centennials * 3155673600ULL;

		quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
		if (quadrennials > 24)
		{
			quadrennials = 24;
		}
		sec -= quadrennials * 126230400ULL;

		annuals = (uint)(sec / 31536000); // 365*24*3600
		if (annuals > 3)
		{
			annuals = 3;
		}
		sec -= annuals * 31536000ULL;

		year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
		leap = !(year % 4) && (year % 100 || !(year % 400));

		// Calculate the day of the year and the time
		yday = sec / 86400;
		sec %= 86400;
		//hour = sec / 3600;
		sec %= 3600;
		//min = sec / 60;
		sec %= 60;

	// Calculate the month
		for (mday = month = 1; month < 13; month++)
		{
			if (yday < daysSinceJan1st[leap][month])
			{
			mday += yday - daysSinceJan1st[leap][month - 1];
			break;
			}
		}
		dest[i] = year*100+month;		
	}
};	


struct gpu_getday
{
	const int_type *source;
    int_type *dest;
		
	gpu_getday(const int_type *_source, int_type *_dest):
			  source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {	
		
		uint64 sec;
		uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
		uint year, leap;
		uint yday, hour, min;
		uint month, mday, wday;
		const uint daysSinceJan1st[2][13]=
		{
			{0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
			{0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
		};
		uint64 SecondsSinceEpoch = source[i]/1000;
		sec = SecondsSinceEpoch + 11644473600;

		wday = (uint)((sec / 86400 + 1) % 7); // day of week
		quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
		sec %= 12622780800ULL;

		centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
		if (centennials > 3)
		{
			centennials = 3;
		}
		sec -= centennials * 3155673600ULL;

		quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
		if (quadrennials > 24)
		{
			quadrennials = 24;
		}
		sec -= quadrennials * 126230400ULL;

		annuals = (uint)(sec / 31536000); // 365*24*3600
		if (annuals > 3)
		{
			annuals = 3;
		}
		sec -= annuals * 31536000ULL;

		year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
		leap = !(year % 4) && (year % 100 || !(year % 400));

		// Calculate the day of the year and the time
		yday = sec / 86400;
		sec %= 86400;
		hour = sec / 3600;
		sec %= 3600;
		min = sec / 60;
		sec %= 60;

	// Calculate the month
		for (mday = month = 1; month < 13; month++)
		{
			if (yday < daysSinceJan1st[leap][month])
			{
			mday += yday - daysSinceJan1st[leap][month - 1];
			break;
			}
		}
		dest[i] = year*10000+month*100+mday;		
	}
};	



void select(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums, queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a,
            CudaSet* b, vector<thrust::device_vector<int_type> >& distinct_tmp, bool& one_liner)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<int_type> exe_nums;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    unsigned int colCount = 0;
    stack<int> col_type;
    string grp_type;
    stack<string> grp_type1;
    stack<string> col_val;
    size_t res_size = 0;

    stack<string> exe_value1;
    stack<int_type*> exe_vectors1;
    stack<float_type*> exe_vectors1_d;
    stack<int_type> exe_nums1;
	stack<unsigned int> exe_precision;
	stack<unsigned int> exe_precision1;
	bool ts;
	stack<bool> exe_ts;
    stack<float_type*> exe_vectors_f;
    stack<float_type> exe_nums_f;
    float_type n1_f, n2_f, res_f;
    bool one_line;
    unsigned int dist_processed = 0;
    bool prep = 0;
    one_line = 0;

    thrust::device_ptr<bool> d_di(thrust::raw_pointer_cast(a->grp.data()));
    std::auto_ptr<ReduceByKeyPreprocessData> ppData;
	
    if (a->grp_count && (a->mRecCount != 0))
        res_size = a->grp_count;
		
	std::clock_t start1 = std::clock();	

    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
        //cout << ss << endl;

        if(ss.compare("emit sel_name") != 0) {
            grp_type = "NULL";

            if (ss.compare("COUNT") == 0  || ss.compare("SUM") == 0  || ss.compare("AVG") == 0 || ss.compare("MIN") == 0 || ss.compare("MAX") == 0 || ss.compare("DISTINCT") == 0 || ss.compare("YEAR") == 0 || ss.compare("MONTH") == 0 || ss.compare("DAY") == 0) {
			
                if(!prep && a->grp_count) {
                    mgpu::ReduceByKeyPreprocess<float_type>((int)a->mRecCount, thrust::raw_pointer_cast(d_di),
                                                            (bool*)0, head_flag_predicate<bool>(), (int*)0, (int*)0,
                                                            &ppData, *context);
                    prep = 1;
                };


                if(!a->grp_count && ss.compare("YEAR") && ss.compare("MONTH") && ss.compare("DAY")) {
                    one_line = 1;
				};	
				
				
				if (ss.compare("YEAR") == 0) {	
					s1_val = exe_value.top();
                    exe_value.pop();
					exe_type.pop();
					thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
					if(a->ts_cols[s1_val]) {						
						thrust::counting_iterator<unsigned int> begin(0);
						gpu_getyear ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
						thrust::for_each(begin, begin + a->mRecCount, ff);						
						exe_precision.push(0);			
					}
					else {
						thrust::transform(a->d_columns_int[s1_val].begin(), a->d_columns_int[s1_val].begin() + a->mRecCount, thrust::make_constant_iterator(10000), res, thrust::divides<int_type>());
						exe_precision.push(a->decimal_zeroes[s1_val]);			
					};	
                    exe_vectors.push(thrust::raw_pointer_cast(res));
                    exe_type.push("VECTOR");					
				}
				else if (ss.compare("MONTH") == 0) {	
					s1_val = exe_value.top();
                    exe_value.pop();
					exe_type.pop();
					thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
					thrust::counting_iterator<unsigned int> begin(0);
					gpu_getmonth ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
					thrust::for_each(begin, begin + a->mRecCount, ff);						
					exe_precision.push(0);			
                    exe_vectors.push(thrust::raw_pointer_cast(res));
                    exe_type.push("VECTOR");					
				}				
				else if (ss.compare("DAY") == 0) {	
					s1_val = exe_value.top();
                    exe_value.pop();
					exe_type.pop();
					thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
					thrust::counting_iterator<unsigned int> begin(0);
					gpu_getday ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
					thrust::for_each(begin, begin + a->mRecCount, ff);						
					exe_precision.push(0);			
                    exe_vectors.push(thrust::raw_pointer_cast(res));
                    exe_type.push("VECTOR");					
				}
				else if (ss.compare("DISTINCT") == 0) {
                    s1_val = exe_value.top();
                    exe_type.pop();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::copy(a->d_columns_int[s1_val].begin(), a->d_columns_int[s1_val].begin() + a->mRecCount,
                                     distinct_tmp[dist_processed].begin());
                        dist_processed++;
                        thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(res_size);
                        exe_vectors.push(thrust::raw_pointer_cast(res));
                        exe_type.push("VECTOR");
                    }
                    else if(a->type[s1_val] == 2) {
                        //will add a DISTINCT on strings if anyone needs it
                        cout << "DISTINCT on strings is not supported yet" << endl;
                        exit(0);
                    }
                    else {
                        cout << "DISTINCT on float is not supported yet" << endl;
                        exit(0);
                    };
                }

                else if (ss.compare("COUNT") == 0) {

                    s1 = exe_type.top();
                    if(s1.compare("VECTOR") != 0) {  // non distinct

                        grp_type = "COUNT";
                        exe_type.pop();
                        s1_val = exe_value.top();
                        exe_value.pop();


                        if (a->grp_count) {
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
							if(alloced_mem.empty()) {		
								alloc_pool(a->maxRecs);
							};
							thrust::device_ptr<int_type> const_seq((int_type*)alloced_mem.back());								
                            thrust::fill(const_seq, const_seq+a->mRecCount, (int_type)1);
                            ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(const_seq), (int_type)0,
                                             mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);

                            //thrust::device_free(const_seq);
                            //thrust::reduce_by_key(d_di, d_di+(a->mRecCount), thrust::constant_iterator<int_type>(1),
                            //                      thrust::make_discard_iterator(), count_diff,
                            //                      head_flag_predicate<bool>(),thrust::plus<int_type>());

                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        }
                        else {
                            thrust::device_ptr<int_type> dest  = thrust::device_malloc<int_type>(1);
                            dest[0] = a->mRecCount;
                            exe_vectors.push(thrust::raw_pointer_cast(dest));
                            exe_type.push("VECTOR");
                        };						
                    }
                    else
                        grp_type = "COUNTD";
					exe_precision.push(0);	

                }
                else if (ss.compare("SUM") == 0) {
				
                    /*if(op_case) {
                    	cout << "found case " << endl;
                    	op_case = 0;
                    	while(!exe_type.empty())
                    	{
                    	cout << "CASE type " << exe_type.top() << endl;
                    	exe_type.pop();
                    	exit(0);
                    	}

                    };
                    */

                    grp_type = "SUM";
                    s1 = exe_type.top();
                    exe_type.pop();

                    if (s1.compare("VECTOR F") == 0) {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();

                        if (a->grp_count) {
                            thrust::device_ptr<float_type> source((float_type*)(s3));
                            thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                            ReduceByKeyApply(*ppData, s3, (float_type)0,
                                             mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);
                            //thrust::reduce_by_key(d_di, d_di + a->mRecCount, source,
                            //                      thrust::make_discard_iterator(), count_diff,
                            //                      head_flag_predicate<bool>(),thrust::plus<float_type>());

                            exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR F");
                        }
                        else {
                            thrust::device_ptr<float_type> source((float_type*)(s3));
                            thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(1);
                            count_diff[0] = mgpu::Reduce(thrust::raw_pointer_cast(source), a->mRecCount, *context);
                            exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR F");
                            a->mRecCount = 1;
                        };
                        hipFree(s3);
                    }
                    if (s1.compare("VECTOR") == 0) {

                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();

                        if (a->grp_count) {
                            thrust::device_ptr<int_type> source((int_type*)(s3));
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                            ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(source), (int_type)0,
                                             mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        }
                        else {
                            thrust::device_ptr<int_type> source((int_type*)(s3));
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(1);
                            count_diff[0] = mgpu::Reduce(thrust::raw_pointer_cast(source), a->mRecCount, *context);
                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        };
                        hipFree(s3);
                    }

                    else if (s1.compare("NAME") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
						
                        if (a->grp_count) {

                            if(a->type[s1_val] == 0) {
                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                                 mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                exe_type.push("VECTOR");
                            }
                            else if(a->type[s1_val] == 1) {
                                thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                                ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                                                 mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);
                                exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                                exe_type.push("VECTOR F");
                            }
                        }
                        else {
                            if(a->type[s1_val] == 0) {
                                thrust::device_ptr<int_type> dest;
                                int_type cc = mgpu::Reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, *context);
                                if (one_line) {
                                    dest = thrust::device_malloc<int_type>(1);
                                    dest[0] = cc;
                                }
                                else {
                                    dest = thrust::device_malloc<int_type>(a->mRecCount);
                                    thrust::sequence(dest, dest+(a->mRecCount), cc, (int_type)0);
                                };
                                exe_vectors.push(thrust::raw_pointer_cast(dest));
                                exe_type.push("VECTOR");
                            }
                            else if(a->type[s1_val] == 1) {
                                thrust::device_ptr<float_type> dest;
                                float_type cc = mgpu::Reduce(thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), a->mRecCount, *context);

                                if (one_line) {
                                    dest = thrust::device_malloc<float_type>(1);
                                    dest[0] = cc;
                                }
                                else {
                                    dest = thrust::device_malloc<float_type>(a->mRecCount);
                                    thrust::sequence(dest, dest+a->mRecCount, cc, (float_type)0);
                                };
                                exe_vectors_f.push(thrust::raw_pointer_cast(dest));
                                exe_type.push("VECTOR F");
                            };
                        };
						exe_precision.push(a->decimal_zeroes[s1_val]);
                    }
                }
                else if (ss.compare("MIN") == 0) {
				
                    grp_type = "MIN";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();
					
					if(alloced_mem.empty()) {								
						alloc_pool(a->maxRecs);
					};
					thrust::device_ptr<unsigned int> d_di1((unsigned int*)alloced_mem.back());								
						
					thrust::copy(d_di, d_di+a->mRecCount,d_di1);
					thrust::exclusive_scan(d_di1, d_di1+a->mRecCount, d_di1);
					thrust::equal_to<unsigned int> binary_pred;					  
					

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        //ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                        //                 mgpu::minimum<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
						thrust::reduce_by_key(d_di1, d_di1+a->mRecCount, a->d_columns_int[s1_val].begin(),
                                              thrust::make_discard_iterator(), count_diff,
                                              binary_pred, thrust::minimum<int_type>());                        
                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");

                    }
                    else if(a->type[s1_val] == 1) {			

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        //ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                        	//					mgpu::minimum<float_type>(), thrust::raw_pointer_cast(count_diff), *context);

						
                        thrust::reduce_by_key(d_di1, d_di1+a->mRecCount, a->d_columns_float[s1_val].begin(),
                                              thrust::make_discard_iterator(), count_diff,
                                              binary_pred, thrust::minimum<float_type>());
                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
					exe_precision.push(a->decimal_zeroes[s1_val]);
                }
                else if (ss.compare("MAX") == 0) {

                    grp_type = "MAX";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                         mgpu::maximum<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");

                    }
                    else if(a->type[s1_val] == 1) {

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                                         mgpu::maximum<float_type>(), thrust::raw_pointer_cast(count_diff), *context);


                        //thrust::reduce_by_key(d_di, d_di+(a->mRecCount), a->d_columns_float[s1_val].begin(),
                        //                      thrust::make_discard_iterator(), count_diff,
                        //                      head_flag_predicate<bool>(), thrust::maximum<float_type>());

                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
					exe_precision.push(a->decimal_zeroes[s1_val]);
                }

                else if (ss.compare("AVG") == 0) {
                    grp_type = "AVG";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                         mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);

                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");
                    }
                    else if(a->type[s1_val] == 1) {

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                                         mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);
                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
					exe_precision.push(a->decimal_zeroes[s1_val]);
                };
            };

            if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("FLOAT") == 0 || ss.compare("VECTOR") == 0 || ss.compare("VECTOR F") == 0) {

                exe_type.push(ss);
                if (ss.compare("NUMBER") == 0) {
                    exe_nums.push(op_nums.front());
                    op_nums.pop();
					exe_precision.push(op_nums_precision.front());
					op_nums_precision.pop();
                }
                if (ss.compare("FLOAT") == 0) {
                    exe_nums_f.push(op_nums_f.front());
                    op_nums_f.pop();
                }
                else if (ss.compare("NAME") == 0) {
                    exe_value.push(op_value.front());
					ts = a->ts_cols[op_value.front()];
                    op_value.pop();					
                }
            }
            else {
                if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                    // get 2 values from the stack
                    s1 = exe_type.top();
                    exe_type.pop();
                    s2 = exe_type.top();
                    exe_type.pop();

                    if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();
                        n2 = exe_nums.top();
                        exe_nums.pop();
						
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();					
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						if(p1) 
							n1 = n1*(unsigned int)pow(10,p1);
						if(p2) 
							n2 = n2*(unsigned int)pow(10,p2);

                        if (ss.compare("ADD") == 0 )
                            res = n1+n2;
                        else if (ss.compare("MUL") == 0 )
                            res = n1*n2;
                        else if (ss.compare("DIV") == 0 )
                            res = n1/n2;
                        else
                            res = n1-n2;

                        thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                        thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                        exe_type.push("VECTOR");
                        exe_vectors.push(thrust::raw_pointer_cast(p));
                    }
                    else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();
                        n2_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        if (ss.compare("ADD") == 0 )
                            res_f = n1_f+n2_f;
                        else if (ss.compare("MUL") == 0 )
                            res_f = n1_f*n2_f;
                        else if (ss.compare("DIV") == 0 )
                            res_f = n1_f/n2_f;
                        else
                            res_f = n1_f-n2_f;

                        thrust::device_ptr<float_type> p = thrust::device_malloc<float_type>(a->mRecCount);
                        thrust::sequence(p, p+(a->mRecCount),res_f,(float_type)0);

                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(thrust::raw_pointer_cast(p));

                    }

                    else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        exe_type.push("VECTOR F");

                        if (a->type[s1_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,1));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s1_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,1));
                        };

                    }
                    else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();

                        exe_type.push("VECTOR F");

                        if (a->type[s2_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,0));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,0));
                        };
                    }
                    else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {

                        s1_val = exe_value.top();
                        exe_value.pop();
                        n1 = exe_nums.top();
                        exe_nums.pop();
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = a->decimal_zeroes[s1_val];					


                        if (a->type[s1_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,(float_type)n1,ss,1));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s1_val);							
							auto pres = precision_func(p2, p1, ss);	
							exe_precision.push(pres);
                            exe_type.push("VECTOR");						
                            exe_vectors.push(a->op(t,n1,ss,1, p2, p1));
                        };
                    }
                    else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = a->decimal_zeroes[s2_val];					

                        if (a->type[s2_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,(float_type)n1,ss,0));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s2_val);
							auto pres = precision_func(p2, p1, ss);	
							exe_precision.push(pres);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,n1,ss,0, p2, p1));
                        };
                    }
                    else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();
						

                        if (a->type[s1_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s1_val);
                            if (a->type[s2_val] == 0) {
                                int_type* t = a->get_int_by_name(s2_val);
								auto p1 = a->decimal_zeroes[s1_val];					
								auto p2 = a->decimal_zeroes[s2_val];					
								auto pres = precision_func(p1, p2, ss);	
								exe_precision.push(pres);

                                exe_type.push("VECTOR");
                                exe_vectors.push(a->op(t,t1,ss,0,p2,p1));
                            }
                            else {
                                float_type* t = a->get_float_type_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            };
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            if (a->type[s2_val] == 0) {
                                int_type* t1 = a->get_int_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            }
                            else {
                                float_type* t1 = a->get_float_type_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            };
                        }
                    }
                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                        s2_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);

                            if (s1.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR");
								auto p1 = exe_precision.top();
								exe_precision.pop();
								auto p2 = a->decimal_zeroes[s2_val];					
								auto pres = precision_func(p1, p2, ss);	
								exe_precision.push(pres);
                                exe_vectors.push(a->op(t,s3,ss,0,p2,p1));
								alloced_mem.push_back(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,0));
								alloced_mem.push_back(s3);
                            }
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            if (s1.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(s3,t, ss,0));
								alloced_mem.push_back(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,0));
								alloced_mem.push_back(s3);
                            }
                        };
                    }
                    else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                        s1_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s1_val] == 0) {
                            int_type* t = a->get_int_by_name(s1_val);

                            if (s2.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR");
								auto p1 = exe_precision.top();
								exe_precision.pop();
								auto p2 = a->decimal_zeroes[s1_val];					
								auto pres = precision_func(p1, p2, ss);	
								exe_precision.push(pres);
                                exe_vectors.push(a->op(t,s3,ss,1,p2,p1));
								alloced_mem.push_back(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,1));
								alloced_mem.push_back(s3);
                            }
                        }
                        else {

                            float_type* t = a->get_float_type_by_name(s1_val);
                            if (s2.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(s3,t,ss,1));
								alloced_mem.push_back(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,1));
								alloced_mem.push_back(s3);
                            }
                        };
                    }
                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto p1 = exe_precision.top();
							exe_precision.pop();
							auto p2 = exe_precision.top();
							exe_precision.pop();							
							auto pres = precision_func(p1, p2, ss);	
							exe_precision.push(pres);
                            exe_vectors.push(a->op(s3,n1, ss,1, p1, p2));
                            //hipFree(s3);
							alloced_mem.push_back(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,(float_type)n1, ss,1));
							alloced_mem.push_back(s3);
                        }
                    }
                    else if (s1.compare("NUMBER") == 0 && (s2.compare("VECTOR") || s2.compare("VECTOR F") == 0)) {
                        n1 = exe_nums.top();
                        exe_nums.pop();

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto p1 = exe_precision.top();
							exe_precision.pop();
							auto p2 = exe_precision.top();
							exe_precision.pop();							
							auto pres = precision_func(p2, p1, ss);	
							exe_precision.push(pres);	
                            exe_vectors.push(a->op(s3,n1, ss,0, p2, p1));
							alloced_mem.push_back(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,(float_type)n1, ss,0));
							alloced_mem.push_back(s3);
                        }
                    }

                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {

                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,1));
							alloced_mem.push_back(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,1));
							alloced_mem.push_back(s3);
                        }
                    }
                    else if (s1.compare("FLOAT") == 0 && (s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0)) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,0));
							alloced_mem.push_back(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,0));
							alloced_mem.push_back(s3);
                        }
                    }

                    else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        int_type* s4 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);	
                        exe_vectors.push(a->op(s3, s4,ss,0,p1,p2));
						alloced_mem.push_back(s3);
						alloced_mem.push_back(s4);
                    }
                    else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,1));
						alloced_mem.push_back(s3);
						alloced_mem.push_back(s4);
                    }
                    else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,0));
						alloced_mem.push_back(s3);
						alloced_mem.push_back(s4);
                    }
                    else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,1));
						alloced_mem.push_back(s3);
						alloced_mem.push_back(s4);
                    }
                }
            }

        } //
        else {
            // here we need to save what is where
			
            col_val.push(op_value.front());
            op_value.pop();
            grp_type1.push(grp_type);

            if(!exe_nums.empty()) {  //number
                col_type.push(0);
                exe_nums1.push(exe_nums.top());
                exe_nums.pop();
				exe_precision1.push(exe_precision.top());				
				exe_precision.pop();
            };
            if(!exe_value.empty()) {  //field name
                col_type.push(1);
				exe_precision1.push(a->decimal_zeroes[exe_value.top()]);
                exe_value1.push(exe_value.top());
				exe_ts.push(ts);				
                exe_value.pop();												
            };
            if(!exe_vectors.empty()) {  //vector int
                exe_vectors1.push(exe_vectors.top());
                exe_vectors.pop();
                col_type.push(2);
				exe_precision1.push(exe_precision.top());
				exe_precision.pop();
            };
            if(!exe_vectors_f.empty()) {  //vector float
                exe_vectors1_d.push(exe_vectors_f.top());
                exe_vectors_f.pop();
                col_type.push(3);
            };

            colCount++;
        };
    };
	
		
    for(unsigned int j=0; j < colCount; j++) {	
	    if ((grp_type1.top()).compare("COUNT") == 0 )
            b->grp_type[col_val.top()] = 0;
        else if ((grp_type1.top()).compare("AVG") == 0 )
            b->grp_type[col_val.top()] = 1;
        else if ((grp_type1.top()).compare("SUM") == 0 )
            b->grp_type[col_val.top()] = 2;
        else if ((grp_type1.top()).compare("NULL") == 0 )
            b->grp_type[col_val.top()] = 3;
        else if ((grp_type1.top()).compare("MIN") == 0 )
            b->grp_type[col_val.top()] = 4;
        else if ((grp_type1.top()).compare("MAX") == 0 )
            b->grp_type[col_val.top()] = 5;
        else if ((grp_type1.top()).compare("COUNTD") == 0 ) {
            b->grp_type[col_val.top()] = 6;
        };
		

        if(col_type.top() == 0) {
            // create a vector
            if (a->grp_count) {
                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                thrust::copy_if(thrust::make_constant_iterator((int)exe_nums1.top()), thrust::make_constant_iterator((int)exe_nums1.top()) + a->mRecCount, d_di, count_diff, thrust::identity<bool>());
                b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) , col_val.top(), res_size);
                thrust::device_free(count_diff);
            }
            else {
				thrust::device_ptr<int_type> s = thrust::device_malloc<int_type>(a->mRecCount);
				thrust::sequence(s, s+(a->mRecCount), (int)exe_nums1.top(), 0);
				b->addDeviceColumn(thrust::raw_pointer_cast(s), col_val.top(), a->mRecCount);
			}	
            exe_nums1.pop();		
			b->decimal_zeroes[col_val.top()] = exe_precision1.top();
			exe_precision1.pop();		
			
        }
        else if(col_type.top() == 1) {
		
            if(a->type[exe_value1.top()] == 0 || a->type[exe_value1.top()] == 2) {
			
                //modify what we push there in case of a grouping
                if (a->grp_count) {
                    thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                    if(!exe_ts.top()) {
						thrust::copy_if(a->d_columns_int[exe_value1.top()].begin(),a->d_columns_int[exe_value1.top()].begin() + a->mRecCount,
										d_di, count_diff, thrust::identity<bool>());
					}					
					else {				
						
						thrust::device_vector<unsigned int> dd_tmp(res_size);
						thrust::copy_if(rcol_matches.begin(), rcol_matches.end(), d_di, dd_tmp.begin(), thrust::identity<bool>());
						thrust::gather(dd_tmp.begin(), dd_tmp.end(), rcol_dev.begin(), count_diff);														
										
					};
                    b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) ,  col_val.top(), res_size);
                    thrust::device_free(count_diff);
                }
                else
                    b->addDeviceColumn(thrust::raw_pointer_cast(a->d_columns_int[exe_value1.top()].data()) , col_val.top(), a->mRecCount);
					
				if(a->type[exe_value1.top()] == 0) {
					b->decimal_zeroes[col_val.top()] = exe_precision1.top();
					b->ts_cols[col_val.top()] = exe_ts.top();
				};
					
                if(a->type[exe_value1.top()] == 2 || (a->type[exe_value1.top()] == 0 && a->string_map.find(exe_value1.top()) != a->string_map.end())) {
                    b->string_map[col_val.top()] = a->string_map[exe_value1.top()];
                };
				exe_precision1.pop();		
				exe_ts.pop();					
            }
            else if(a->type[exe_value1.top()] == 1) {
                //modify what we push there in case of a grouping
                if (a->grp_count) {
                    thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                    //thrust::device_ptr<bool> d_grp(a->grp);
                    thrust::copy_if(a->d_columns_float[exe_value1.top()].begin(), a->d_columns_float[exe_value1.top()].begin() + a->mRecCount,
                                    d_di, count_diff, thrust::identity<bool>());
                    b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) , col_val.top(), res_size, a->decimal[exe_value1.top()]);
                    thrust::device_free(count_diff);
                }
                else {
                    b->addDeviceColumn(thrust::raw_pointer_cast(a->d_columns_float[exe_value1.top()].data()), col_val.top(), a->mRecCount, a->decimal[exe_value1.top()]);
				};	
            }
            exe_value1.pop();	
        }
        else if(col_type.top() == 2) {	    // int

            if (a->grp_count)
                b->addDeviceColumn(exe_vectors1.top() , col_val.top(), res_size);
            else {
                if(!one_line)
                    b->addDeviceColumn(exe_vectors1.top() , col_val.top(), a->mRecCount);
                else
                    b->addDeviceColumn(exe_vectors1.top() , col_val.top(), 1);
            };
            hipFree(exe_vectors1.top());
            exe_vectors1.pop();
			b->decimal_zeroes[col_val.top()] = exe_precision1.top();
			exe_precision1.pop();		
			
        }
        else if(col_type.top() == 3) {        //float
		
            if (a->grp_count) {
                b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), res_size, 1);
            }
            else {
                if(!one_line) {
                    b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), a->mRecCount, 1);
                }
                else {
                    b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), 1, 1);
                };
            };
            hipFree(exe_vectors1_d.top());
            exe_vectors1_d.pop();
        };
        col_type.pop();
        col_val.pop();
        grp_type1.pop();
    };


    if (!a->grp_count) {
        if(!one_line)
            b->mRecCount = a->mRecCount;
        else
            b->mRecCount = 1;
        one_liner = one_line;
    }
    else {
        b->mRecCount = res_size;
        one_liner = 0;
    };
}


