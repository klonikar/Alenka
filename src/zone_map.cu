#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "zone_map.h"

void process_error(int severity, string err);   // this should probably live in a utils header file

bool fh_equal_to(const float_type x, const float_type y)
{
    return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
}


bool fh_less(const float_type x, const float_type y)
{
    return ((y-x) > EPSILON);
}


bool fh_greater(const float_type x, const float_type y)
{
    return ((x-y) > EPSILON);
}


bool fh_greater_equal_to(const float_type x, const float_type y)
{
    return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}


bool fh_less_equal_to(const float_type x, const float_type y)
{
    return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}

char host_logical_and(char column1, char column2)
{
    //cout << "AND " << column1 << " " << column2 << endl;
    if (column1 == 'A' && column2 == 'A')
        return 'A';
    else if (column1 == 'N' || column2 == 'N') {
        return 'N';
    }
    else
        return 'R';

}


char host_logical_or(char column1, char column2)
{
    //cout << "OR " << column1 << " " << column2 << endl;
    if (column1 == 'A' && column2 == 'A')
        return 'A';
    else if (column1 == 'N' && column2 == 'N')
        return 'N';
    else
        return 'R';

}



char host_compare(int_type s, int_type d, int_type op_type)
{
    char res = 'N';

    if (op_type == 2 && d>s ) // >
        res = 'A';
    else if (op_type == 1 && d<s)  // <
        res = 'A';
    else if (op_type == 6 && d>=s) // >=
        res = 'A';
    else if (op_type == 5 && d<=s)  // <=
        res = 'A';
    else if (op_type == 4 && d==s)// =
        res = 'A';
    else // !=
        if(d!=s) res = 'A';

    return res;
}


char host_compare(float_type s, float_type d, int_type op_type)
{
    char res = 'N';

    if (op_type == 2 && (d-s) > EPSILON) // >
        res = 'A';
    else if (op_type == 1 && (s-d) > EPSILON)  // <
        res = 'A';
    else if (op_type == 6 && ((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) // >=
        res = 'A';
    else if (op_type == 5 && ((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON)))  // <=
        res = 'A';
    else if (op_type == 4 && ((d-s) < EPSILON) && ((d-s) > -EPSILON))// =
        res = 'A';
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 'A';

    return res;
}


char host_compare(int_type* column1, int_type d, int_type op_type)
{
    char res = 'R';
	
	if (op_type == 2) {   // >
        if (column1[1] <= d)
            res = 'N';
        else if (column1[0] > d)
            res = 'A';
    }
    else if (op_type == 1) { // <
        if (column1[0] >= d)
            res = 'N';
        else if (column1[1] < d)
            res = 'A';
    }
    else if (op_type == 6) {  // >=
        if (column1[1] < d)
            res = 'N';
        else if (column1[0] >= d)
            res = 'A';
    }
    else if (op_type == 5) { // <=
        if (column1[0] > d)
            res = 'N';
        else if (column1[1] <= d)
            res = 'A';
    }
    else if (op_type == 4 && column1[0] == d && column1[1] == d) { // =
        res = 'A';
    };
    //cout << "res " << res << endl;

    return res;
}

char host_compare(float_type* column1, float_type d, int_type op_type)
{
    char res = 'R';
    //cout << "CMP " << column1[0] << " " << column1[1] << " with " << d << endl;

    if (op_type == 2) { // >
        if(fh_less_equal_to(column1[1],d)) {
            res = 'N';
        }
        else if(fh_greater(column1[0],d)) {
            res = 'A';
        };
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],d)) {
            res = 'A';
        }
        else if(fh_greater_equal_to(column1[0],d)) {
            res = 'N';
        };
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[0],d)) {
            res = 'A';
        }
        else if(fh_less(column1[1],d)) {
            res = 'N';
        };
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],d)) {
            res = 'A';
        }
        else if(fh_greater(column1[0],d)) {
            res = 'N';
        };
    }
    else if (op_type == 4 && fh_equal_to(column1[0],d) && fh_equal_to(column1[1],d)) // =
        res = 'A';

    //cout << "res " << res << endl;
    return res;

}


char host_compare(int_type* column1, int_type* column2, int_type op_type)
{
    char res = 'R';	
	
    if (op_type == 2) { // >
        if(column1[0] > column2[1])
            res = 'A';
        else if(column1[1] <= column2[0])
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(column1[1] < column2[0])
            res = 'A';
        else if(column1[0] >= column2[1])
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(column1[0] >= column2[1])
            res = 'A';
        else if(column1[1] < column2[0])
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(column1[1] <= column2[0])
            res = 'A';
        else if(column1[0] > column2[1])
            res = 'N';
    }
    else if (op_type == 4  && column1[0] == column2[1] && column1[1] == column2[0]) // =
        res = 'A';

    return res;


}

char host_compare(float_type* column1, float_type* column2, int_type op_type)
{
    char res = 'R';

    if (op_type == 2) { // >
        if(fh_greater(column1[0],column2[1]))
            res = 'A';
        else if(fh_less_equal_to(column1[1],column2[0]))
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],column2[0]))
            res = 'A';
        else if(fh_greater_equal_to(column1[0],column2[1]))
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[1],column2[0]))
            res = 'A';
        else if(fh_less(column1[1],column2[0]))
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],column2[0]))
            res = 'A';
        else if(fh_greater(column1[0],column2[1]))
            res = 'N';
    }
    else if (op_type == 4  && fh_equal_to(column1[0], column2[1]) && fh_equal_to(column1[1],column2[0])) // =
        res = 'A';

    return res;
}


char host_compare(float_type* column1, int_type* column2, int_type op_type)
{
    char res = 'R';

    if (op_type == 2) { // >
        if(fh_greater(column1[0],(float_type)column2[1]))
            res = 'A';
        else if(fh_less_equal_to(column1[1],(float_type)column2[0]))
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_greater_equal_to(column1[0],(float_type)column2[1]))
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_less(column1[1],(float_type)column2[0]))
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_greater(column1[0],(float_type)column2[1]))
            res = 'N';
    }
    else if (op_type == 4  && fh_equal_to(column1[0],(float_type) column2[1]) && fh_equal_to(column1[1],(float_type)column2[0])) // =
        res = 'A';

    return res;
}



float_type* host_op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    float_type* temp = (float_type*)malloc(2*float_size);
    temp[0] = (float_type)column1[0];
    temp[1] = (float_type)column1[1];

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = temp[0] * column2[0];
            temp[1] = temp[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = temp[0] + column2[0];
            temp[1] = temp[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - temp[0];
            temp[1] = column2[1] - temp[1];
        }
        else {
            temp[0] = column2[0] / temp[0];
            temp[1] = column2[1] / temp[1];
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = temp[0] * column2[0];
            temp[1] = temp[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = temp[0] + column2[0];
            temp[1] = temp[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = temp[0] - column2[0];
            temp[1] = temp[1] - column2[1];
        }
        else {
            temp[0] = temp[0] / column2[0];
            temp[1] = temp[1] / column2[1];
        }
    };

    return temp;
}




int_type* host_op(int_type* column1, int_type* column2, string op_type, int reverse)
{
    int_type* temp = (int_type*)malloc(2*int_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - column2[0];
            temp[1] = column1[1] - column2[1];
        }
        else {
            temp[0] = column1[0] / column2[0];
            temp[1] = column1[1] / column2[1];
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - column1[0];
            temp[1] = column2[1] - column1[1];
        }
        else {
            temp[0] = column2[0] / column1[0];
            temp[1] = column2[1] / column1[1];
        }
    }

    return temp;

}

float_type* host_op(float_type* column1, float_type* column2, string op_type, int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - column2[0];
            temp[1] = column1[1] - column2[1];
        }
        else {
            temp[0] = column1[0] / column2[0];
            temp[1] = column1[1] / column2[1];
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - column1[0];
            temp[1] = column2[1] - column1[1];
        }
        else {
            temp[0] = column2[0] / column1[0];
            temp[1] = column2[1] / column1[1];
        }
    }

    return temp;

}

int_type* host_op(int_type* column1, int_type d, string op_type, int reverse)
{
    int_type* temp = (int_type*)malloc(2*int_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - d;
            temp[1] = column1[1] - d;
        }
        else {
            temp[0] = column1[0] / d;
            temp[1] = column1[1] / d;
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = d - column1[0];
            temp[1] = d - column1[1];
        }
        else {
            temp[0] = d / column1[0];
            temp[1] = d / column1[1];
        }

    };
    return temp;

}

float_type* host_op(int_type* column1, float_type d, string op_type, int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);
    temp[0] = (float_type)column1[0];
    temp[1] = (float_type)column1[1];

    float_type* temp1 = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp1[0] = temp[0] * d;
            temp1[1] = temp[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp1[0] = temp[0] + d;
            temp1[1] = temp[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp1[0] = temp[0] - d;
            temp1[1] = temp[1] - d;
        }
        else {
            temp1[0] = temp[0] / d;
            temp1[1] = temp[1] / d;
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp1[0] = temp[0] * d;
            temp1[1] = temp[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp1[0] = temp[0] + d;
            temp1[1] = temp[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp1[0] = d - temp[0];
            temp1[1] = d - temp[1];
        }
        else {
            temp1[0] = d / temp[0];
            temp1[1] = d / temp[1];
        }
    };

    free(temp);
    return temp1;

}



float_type* host_op(float_type* column1, float_type d, string op_type,int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - d;
            temp[1] = column1[1] - d;
        }
        else {
            temp[0] = column1[0] / d;
            temp[1] = column1[1] / d;
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = d - column1[0];
            temp[1] = d - column1[1];
        }
        else {
            temp[0] = d / column1[0];
            temp[1] = d / column1[1];
        }
    };

    return temp;

}


unsigned int precision_func(unsigned int& p1, unsigned int& p2, string op) {

	if (op.compare("DIV") != 0 ) {
		unsigned int res;
		if (op.compare("MUL") != 0 ) {			
			if(p1 > p2) {
				res = p1;				
				p2 = p1-p2;				
				p1 = 0;
			}	
			else {
				res = p1;				
				p1 = p2-p1;				
				p2 = 0; 
			};				
			return res;
		}	
		else {
			//std::swap(p1,p2);
			res = p1+p2;
			p1 = 0;
			p2 = 0;
			return res;
		};	
	}
	else {
		if(p1 == p2) {
			p1 = p1+4;			
			p2 = 0;
			return p1;				
		}
		else {
			if(p1 > p2) {
				p1 = p1 + (p1-p2) + 4;
				p2 = 0;
				return p1;
			}
			else {
				p2 = p2 + (p2-p1) + 4;
				p1 = 0;
				return p2;
			}						
		}		
	};
}



//CudaSet a contains two records - with all minimum and maximum values of the segment
//We need to determine if this segment needs to be processed
//The check takes place in host's memory

char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a, unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<float_type*> exe_vectors_f;
    stack<int_type> exe_nums;
    stack<char> bool_vectors;
    stack<float_type> exe_nums_f;
	stack<unsigned int> exe_precision;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    float_type n1_f, n2_f, res_f;

    if(a->not_compressed)
        return 'R';

    //first we need to set all host arrays [0] and [1] of t to min and max values of appropriate files
    set<string> uniques;
    queue<string> fields(op_value);
    CudaSet *t;
    FILE* f;
    unsigned int cnt;
    string f1;


    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{

            if(a->filtered)
                t = varNames[a->source_name];
            else
                t = a;

            // copy t min and max values to a only if int, decimal or float
            if(t->type[fields.front()] <= 1) {

                f1 = t->load_file_name + "." + fields.front() + "." + to_string(segment);
                f = fopen (f1.c_str() , "rb" );
                if(!f) {
                    cout << "Error opening " << f1 << " file " << endl;
                    exit(0);
                };

                fread((char *)&cnt, 4, 1, f);
                if (t->type[fields.front()] == 0) {
                    a->h_columns_int[fields.front()].resize(2);
                    fread((char *)&a->h_columns_int[fields.front()][0], 8, 1, f);
                    fread((char *)&a->h_columns_int[fields.front()][1], 8, 1, f);
					fseek(f, 8+cnt, SEEK_CUR);
					fread((char *)&a->mRecCount, 4, 1, f);
					//cout << endl << "ZONE " << a->mRecCount << endl;
					fread((char *)&cnt, 4, 1, f);					
                    //cout << "file " << f1 << " " << segment << " " << a->h_columns_int[fields.front()][0] << ":" << a->h_columns_int[fields.front()][1] << endl;
                }
                else  {
                    long long int t;
                    a->h_columns_float[fields.front()].resize(2);
                    fread((char *)&t, 8, 1, f);
                    a->h_columns_float[fields.front()][0] = (float_type)t/100.0;
                    fread((char *)&t, 8, 1, f);
                    a->h_columns_float[fields.front()][1] = (float_type)t/100.0;
                    //cout << "file " << f1 << " " << segment << " " << a->h_columns_float[a->type_index[colIndex]][0] << ":" << a->h_columns_float[a->type_index[colIndex]][1] << endl;
                };
                fclose(f);
            };
        };
        uniques.insert(fields.front());
        fields.pop();
    };


    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
		//cout << ss << endl;

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0) {

            exe_type.push(ss);
            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
				exe_precision.push(op_nums_precision.front());
				op_nums_precision.pop();
            }
            else if (ss.compare("NAME") == 0)  {
                if(var_exists(a, op_value.front())) {
                    exe_value.push(op_value.front());
                    op_value.pop();
                }
                else {
                    process_error(1, "Couldn't find column " + op_value.front());
                    //cout << "Couldn't find column " << op_value.front() << endl;
                    //exit(0);
                };
            }
            else if (ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
            }
            if (ss.compare("FLOAT") == 0) {
                exe_nums_f.push(op_nums_f.front());
                op_nums_f.pop();
            }

        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack
                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();


                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);
					
					if(p1) 
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,p2);

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;


                    exe_type.push("NUMBER");
                    exe_nums.push(res);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (ss.compare("ADD") == 0 )
                        res_f = n1_f+n2_f;
                    else if (ss.compare("MUL") == 0 )
                        res_f = n1_f*n2_f;
                    else if (ss.compare("DIV") == 0 )
                        res_f = n1_f/n2_f;
                    else
                        res_f = n1_f-n2_f;

                    exe_type.push("FLOAT");
                    exe_nums_f.push(res_f);
                }
				else if (s1.compare("STRING") == 0 && s2.compare("STRING") == 0) { // date() + 3DAY or "1970-01-01 ..." + 2MONTH
				    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val, tt;
					bool reverse = 1;

					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						tt = curr_time;		
					}
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
						}
						else {							
							reverse = 0;
							pos = s1_val.find("date()");
							if(pos != string::npos) {
								tt = curr_time;		
							}
							else {
								pos = s1_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s1_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s1_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s1_val.substr(8,2));
									tm.tm_hour = std::stoi(s1_val.substr(11,2));
									tm.tm_min = std::stoi(s1_val.substr(14,2));
									tm.tm_sec = std::stoi(s1_val.substr(17,2));								
									#ifdef _WIN64
									tt = _mkgmtime (&tm);
									#else
									tt = timegm (&tm);
									#endif							
									tt = tt*1000 + std::stoi(s1_val.substr(20,3));					
								}
							};							
						};	
					};


					pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = (add_interval(tt/1000, 0, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0) - tt/1000)*1000;
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = (add_interval(tt/1000, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0, 0) - tt/1000)*1000;
										}							
										else {											
											pos = s1_val.find("DAY");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
											}					
											else {
												pos = s1_val.find("HOUR");
												if(pos != string::npos) {
													val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
												}											
												else {
													pos = s1_val.find("MINUTE");
													if(pos != string::npos) {
														val = stoi(s1_val.substr(0, pos)) * 60*1000;								
													}											
													else {
														pos = s1_val.find("MSECOND");
														if(pos != string::npos) {
															val = stoi(s1_val.substr(0, pos));
														}											
														else {
															pos = s1_val.find("MONTH");
															if(pos != string::npos) {
																val = stoi(s1_val.substr(0, pos));
																val = (add_interval(tt/1000, 0, val, 0, 0, 0, 0) - tt/1000)*1000;
															}						
															else {
																pos = s1_val.find("YEAR");
																if(pos != string::npos) {
																	val = stoi(s1_val.substr(0, pos));
																	val = (add_interval(tt/1000, val, 0, 0, 0, 0, 0) - tt/1000)*1000;
																}	
																else {
																	pos = s2_val.find("SECOND");
																	if(pos != string::npos) {
																		val = stoi(s2_val.substr(0, pos))*1000;
																	}	
																	else {
																		pos = s1_val.find("SECOND");
																		if(pos != string::npos) {
																			val = stoi(s1_val.substr(0, pos))*1000;
																		}	
																	}		
																}																		
															};								
														};
													};							
												};
											}											
										}	
									};									
								};
							};
						}	
					};				
	
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+tt;
                    else {
						if(!reverse)
							res = val-tt;
						else
							res = tt- val;
					};	

                    exe_type.push("NUMBER");
                    exe_nums.push(res);		
					exe_precision.push(0);					
				}				
				
				else if (s1.compare("STRING") == 0 && s2.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = val-n1;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				else if (s2.compare("STRING") == 0 && s1.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = n1-val;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				
				else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_host_int_by_name(s1_val);
					
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(host_op(t,val*24*60*60*1000,ss,1));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(host_op(t,val*60*60*1000,ss,1));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(host_op(t,val*60*1000,ss,1));						
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(host_op(t,val,ss,1));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
									    int_type* temp = (int_type*)malloc(2*int_size);
										if (ss.compare("ADD") != 0 )
											val = -val;
										temp[0] = add_interval(t[0], 0, val, 0, 0, 0, 0);
										temp[1] = add_interval(t[1], 0, val, 0, 0, 0, 0);
										exe_vectors.push(temp);
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											temp[0] = add_interval(t[0], val, 0, 0, 0, 0, 0);
											temp[1] = add_interval(t[1], val, 0, 0, 0, 0, 0);
											exe_vectors.push(temp);
										}							
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(host_op(t,val*1000,ss,1));						
											}	
										}	
									};									
								};
							};

						};
					}
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				else if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_host_int_by_name(s2_val);
					//cout << "name " << s2_val << endl;
					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos));
						exe_vectors.push(host_op(t,val*24*60*60*1000,ss,0));
					}
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos));
							exe_vectors.push(host_op(t,val*60*60*1000,ss,0));						
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos));
								exe_vectors.push(host_op(t,val*60*1000,ss,0));						
							}		
							else {
								pos = s1_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s1_val.substr(0, pos));
									exe_vectors.push(host_op(t,val,ss,1));						
								}											
								else {
									pos = s1_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s1_val.substr(0, pos));
									    int_type* temp = (int_type*)malloc(2*int_size);
										if (ss.compare("ADD") != 0 )
											val = -val;
										temp[0] = add_interval(t[0], 0, val, 0, 0, 0, 0);
										temp[1] = add_interval(t[1], 0, val, 0, 0, 0, 0);
										exe_vectors.push(temp);
									}						
									else {
										pos = s1_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s1_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											temp[0] = add_interval(t[0], val, 0, 0, 0, 0, 0);
											temp[1] = add_interval(t[1], val, 0, 0, 0, 0, 0);
											exe_vectors.push(temp);
										}		
										else {
											pos = s1_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos));
												exe_vectors.push(host_op(t,val*1000,ss,1));						
											}	
										}	
										
									};									
								};
							};
							
						};						
					}
					
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}				
				
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,1));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,1));
                    };

                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,0));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s1_val];					

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(t,(float_type)n1,ss,1));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s1_val);
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,p1);
							t[1] = t[1]*(unsigned int)pow(10,p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,p2);
						};	
							
                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(t,n1,ss,1));
                    };
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(t,(float_type)n1,ss,0));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s2_val);
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,p1);
							t[1] = t[1]*(unsigned int)pow(10,p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,p2);
						};	

                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(t,n1,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {

                    return 'R';
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                    s2_val = exe_value.top();
                    exe_value.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto pres = precision_func(p1, p2,ss);	
							exe_precision.push(pres);
							
							if(p1) {
								t[0] = t[0]*(unsigned int)pow(10,p1);
								t[1] = t[1]*(unsigned int)pow(10,p1);
							};	
							if(p2) {
								s3[0] = s3[0]*(unsigned int)pow(10,p2);
								s3[1] = s3[1]*(unsigned int)pow(10,p2);
							};	
							
                            exe_vectors.push(host_op(t,s3,ss,0));
                            delete [] s3;
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,0));
                            delete [] s3;
                        }
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(s3,t, ss,0));
                            delete [] s3;
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,0));
                            delete [] s3;
                        }
                    };
                }
                else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    exe_value.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s1_val];					

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto pres = precision_func(p1, p2, ss);	
							exe_precision.push(pres);
							if(p1) {
								t[0] = t[0]*(unsigned int)pow(10,p1);
								t[1] = t[1]*(unsigned int)pow(10,p1);
							};	
							if(p2) {
								s3[0] = s3[0]*(unsigned int)pow(10,p2);
								s3[1] = s3[1]*(unsigned int)pow(10,p2);
							};	
							
                            exe_vectors.push(host_op(t,s3,ss,1));
                            delete [] s3;
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,1));
                            delete [] s3;
                        }
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(s3,t,ss,1));
                            delete [] s3;
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,1));
                            delete [] s3;
                        }
                    };
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						
						if(p1) {
							s3[0] = s3[0]*(unsigned int)pow(10,p1);
							s3[1] = s3[1]*(unsigned int)pow(10,p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,p2);
						};	
						
                        exe_vectors.push(host_op(s3,n1, ss,1));
                        delete [] s3;
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,(float_type)n1, ss,1));
                        delete [] s3;
                    }
                }
                else if (s1.compare("NUMBER") == 0 &&( s2.compare("VECTOR") || s2.compare("VECTOR F") == 0)) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						if(p1) {
							s3[0] = s3[0]*(unsigned int)pow(10,p1);
							s3[1] = s3[1]*(unsigned int)pow(10,p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,p2);
						};	

                        exe_vectors.push(host_op(s3,n1, ss,0));
                        delete [] s3;
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,(float_type)n1, ss,0));
                        delete [] s3;
                    }
                }

                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,1));
                        delete [] s3;
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,1));
                        delete [] s3;
                    }
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    n1_f = (float_type)exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,0));
                        delete [] s3;
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,0));
                        delete [] s3;
                    }
                }

				else if (s1.compare("VECTOR") == 0 && s2.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    int_type* s4 = exe_vectors.top();
					exe_vectors.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(host_op(s4,val*24*60*60*1000,ss,1));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(host_op(s4,val*60*60*1000,ss,1));
						}
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(host_op(s4,val*60*1000,ss,1));
							}
							else {
								pos = s2_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;							
									exe_vectors.push(host_op(s4,val,ss,1));									
								}
								else {
									pos = s2_val.find("MSECOND");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										exe_vectors.push(host_op(s4,val,ss,1));						
									}											
									else {
										pos = s2_val.find("MONTH");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;
											temp[0] = add_interval(s4[0], 0, val, 0, 0, 0, 0);
											temp[1] = add_interval(s4[1], 0, val, 0, 0, 0, 0);
											exe_vectors.push(temp);
										}						
										else {
											pos = s2_val.find("YEAR");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												int_type* temp = (int_type*)malloc(2*int_size);
												if (ss.compare("ADD") != 0 )
													val = -val;											
												temp[0] = add_interval(s4[0], val, 0, 0, 0, 0, 0);
												temp[1] = add_interval(s4[1], val, 0, 0, 0, 0, 0);
												exe_vectors.push(temp);
											}							
											else {
												pos = s2_val.find("SECOND");
												if(pos != string::npos) {
													val = stoi(s2_val.substr(0, pos));
													exe_vectors.push(host_op(s4,val*1000,ss,1));						
												}											
											};
										}	
									};									
								};								
							}		
						};
					}
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				
				else if (s2.compare("VECTOR") == 0 && s1.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* s4 = exe_vectors.top();
					exe_vectors.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(host_op(s4,val*24*60*60*1000,ss,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(host_op(s4,val*60*60*1000,ss,0));
						}
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(host_op(s4,val*60*1000,ss,0));
							}
							else {
								pos = s2_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;							
									exe_vectors.push(host_op(s4,val,ss,0));									
								}	
								else {
									pos = s2_val.find("MSECOND");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										exe_vectors.push(host_op(s4,val,ss,1));						
									}											
									else {
										pos = s2_val.find("MONTH");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;
											temp[0] = add_interval(s4[0], 0, val, 0, 0, 0, 0);
											temp[1] = add_interval(s4[1], 0, val, 0, 0, 0, 0);
											exe_vectors.push(temp);
										}						
										else {
											pos = s2_val.find("YEAR");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												int_type* temp = (int_type*)malloc(2*int_size);
												if (ss.compare("ADD") != 0 )
													val = -val;											
												temp[0] = add_interval(s4[0], val, 0, 0, 0, 0, 0);
												temp[1] = add_interval(s4[1], val, 0, 0, 0, 0, 0);
												exe_vectors.push(temp);
											}							
											else {
												pos = s2_val.find("SECOND");
												if(pos != string::npos) {
													val = stoi(s2_val.substr(0, pos));
													exe_vectors.push(host_op(s4,val*1000,ss,1));						
												}											
											}
										};									
									};
								};

							}
						};
					}

					
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				
				
                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s4 = exe_vectors.top();
                    exe_vectors.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);
					
					if(p1) {
						s3[0] = s3[0]*(unsigned int)pow(10,p1);
						s3[1] = s3[1]*(unsigned int)pow(10,p1);
					};	
					if(p2) {
						s4[0] = s4[0]*(unsigned int)pow(10,p2);
						s4[1] = s4[1]*(unsigned int)pow(10,p2);
					};						
					
					exe_type.push("VECTOR");
                    exe_vectors.push(host_op(s3, s4,ss,1));
                    delete [] s3;
                    delete [] s4;
                }
                else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,1));
                    delete [] s3;
                    delete [] s4;
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,0));
                    delete [] s3;
                    delete [] s4;
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,1));
                    delete [] s3;
                    delete [] s4;
                }
            }

            else if (ss.compare("CMP") == 0) {

                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();


                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = std::max(p1, p2);	
					exe_precision.push(pres);					
                    exe_type.push("VECTOR");
					if(p1) 
						n1 = n1*(unsigned int)pow(10,pres-p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,pres-p2);
                    bool_vectors.push(host_compare(n1,n2,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,n2_f,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NUMBER") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,float_type(n2),cmp_type));
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,float_type(n2),cmp_type));
                }

                else if (s1.compare("STRING") == 0 && s2.compare("NAME") == 0) {
                    //exe_type.push("VECTOR");
                    //bool_vectors.push('R'); // later I plan to change implementation of char type so I will leave indexing of char off for now
					//only date() can be in STRING					
					time_t tt;
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_host_int_by_name(s2_val);
					
					auto pos = s1_val.find("date()");
					if(pos != string::npos) {
						val = curr_time;
						bool_vectors.push(host_compare(t,val, cmp_type));
					}	
					else {
						pos = s1_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s1_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s1_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s1_val.substr(8,2));
							tm.tm_hour = std::stoi(s1_val.substr(11,2));
							tm.tm_min = std::stoi(s1_val.substr(14,2));
							tm.tm_sec = std::stoi(s1_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s1_val.substr(20,3));					
							bool_vectors.push(host_compare(t,tt, cmp_type));
						}
						else
							bool_vectors.push('R');
					};


					
					exe_type.push("VECTOR");
                }
                else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    //exe_type.push("VECTOR");
                    //bool_vectors.push('R');
                    s2_val = exe_value.top();
                    exe_value.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					time_t tt;
					int_type* t = a->get_host_int_by_name(s1_val);
					
					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						val = curr_time;
						bool_vectors.push(host_compare(t,val, cmp_type));
					}	
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
							bool_vectors.push(host_compare(t,tt, cmp_type));
						}
						else
							bool_vectors.push('R');
					};
					exe_type.push("VECTOR");
                }


                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_type.push("VECTOR");
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);
						
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,pres-p1);
							t[1] = t[1]*(unsigned int)pow(10,pres-p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,pres-p2);
						};							
						
                        bool_vectors.push(host_compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(float_type)n1,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_type.push("VECTOR");
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,pres-p1);
							t[1] = t[1]*(unsigned int)pow(10,pres-p1);
						};	
						if(p2) {
							n1 = n1*(unsigned int)pow(10,pres-p2);
						};							

                        bool_vectors.push(host_compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(float_type)n1,cmp_type));
                    };
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    }
                    else {

                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    };
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(float_type)n1,cmp_type));
                    delete [] s3;
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = std::max(p1, p2);	
					exe_precision.push(pres);
					
					if(p1) {
						s3[0] = s3[0]*(unsigned int)pow(10,pres-p1);
						s3[1] = s3[1]*(unsigned int)pow(10,pres-p1);
					};	
					if(p2) {
						n1 = n1*(unsigned int)pow(10,pres-p2);
					};							
					
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1,cmp_type));
                    delete [] s3;
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR F") == 0) {

                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(float_type)n1,cmp_type));
                    delete [] s3;
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") == 0) {

                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto pres = std::max(p1, p2);	
					exe_precision.push(pres);
					if(p1) {
						s3[0] = s3[0]*(unsigned int)pow(10,pres-p1);
						s3[1] = s3[1]*(unsigned int)pow(10,pres-p1);
					};	
					if(p2) {
						n1 = n1*(unsigned int)pow(10,pres-p2);
					};							
					
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1,cmp_type));
                    delete [] s3;
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1_f,cmp_type));
                    delete [] s3;
                }
                else if (s1.compare("VECTOR") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(int_type)n1_f,cmp_type));
                    delete [] s3;
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1_f,cmp_type));
                    delete [] s3;
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(int_type)n1_f,cmp_type));
                    delete [] s3;
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NAME") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    delete [] s3;
                }
				
				else if (s1.compare("VECTOR") == 0 && s2.compare("STRING") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					exe_precision.pop();
					time_t tt;
					
					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						tt = curr_time;								
					}
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
						}
					};
					
					
					
                    bool_vectors.push(host_compare(s3,tt,cmp_type));
                    delete [] s3;
				}
				
				else if (s2.compare("VECTOR") == 0 && s1.compare("STRING") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					exe_precision.pop();					
					time_t tt;
					
					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						tt = curr_time;								
					}
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
						}
					};

                    bool_vectors.push(host_compare(s3,tt,cmp_type));
                    delete [] s3;
				}


                else if (s1.compare("VECTOR") == 0 && s2.compare("NAME") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					


                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,pres-p1);
							t[1] = t[1]*(unsigned int)pow(10,pres-p1);
						};	
						if(p2) {
							s3[0] = s3[0]*(unsigned int)pow(10,pres-p2);
							s3[1] = s3[1]*(unsigned int)pow(10,pres-p2);
						};							

                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    delete [] s3;
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR F") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    delete [] s3;
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto p1 = a->decimal_zeroes[s2_val];					

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);
						if(p1) {
							t[0] = t[0]*(unsigned int)pow(10,pres-p1);
							t[1] = t[1]*(unsigned int)pow(10,pres-p1);
						};	
						if(p2) {
							s3[0] = s3[0]*(unsigned int)pow(10,pres-p2);
							s3[1] = s3[1]*(unsigned int)pow(10,pres-p2);
						};							

                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    delete [] s3;
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = std::max(p2, p1);	
					exe_precision.push(pres);
					if(p1) {
						s3[0] = s3[0]*(unsigned int)pow(10,pres-p1);
						s3[1] = s3[1]*(unsigned int)pow(10,pres-p1);
					};	
					if(p2) {
						s2[0] = s2[0]*(unsigned int)pow(10,pres-p2);
						s2[1] = s2[1]*(unsigned int)pow(10,pres-p2);
					};							
					
                    bool_vectors.push(host_compare(s2,s3,cmp_type));
                    delete [] s3;
                    delete [] s2;
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s2 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s2,s3,cmp_type));
                    delete [] s3;
                    delete [] s2;
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,s2,cmp_type));
                    delete [] s3;
                    delete [] s2;
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,s2,cmp_type));
                    delete [] s3;
                    delete [] s2;
                }


                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    return 'R';
                }
            }

            else if (ss.compare("AND") == 0) {
                char s3 = bool_vectors.top();
                bool_vectors.pop();
                char s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(host_logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                char s3 = bool_vectors.top();
                bool_vectors.pop();
                char s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(host_logical_or(s2,s3));
            }
            else {
                if(ss.compare("JOIN") == 0)
                    process_error(2, "operation = is not valid");
                //cout << "operation = is not valid" << endl;
                else
                    process_error(2, "operation " + string(ss)+ " is not valid");
                //cout << "operation " << ss << " is not valid" << endl;
                exit(0);	// never gets here
            }
        };
    };


    return bool_vectors.top();

}

