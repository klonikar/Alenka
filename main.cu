#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <map>
#include <string>
#include <iostream>
#include <ctime>
#include <thrust/device_vector.h>
#include "moderngpu/src/moderngpu/kernel_reduce.hxx"
#include "moderngpu/src/moderngpu/kernel_segreduce.hxx"

using namespace std;
#include "alenka.h"

int main(int ac, char **av)
{
    std::clock_t start;
    int x;

    if(ac < 2) {
        cout << "Usage : alenka [--QPS-test] | [ [-l load size(MB)] [-v] script.sql ]" << endl;
        exit(1);	
    }
    // test QPS via alenkaExecute	-- this section is the only C++ dependency
    else if (string(av[1]) == "--QPS-test") {
        alenkaInit(NULL);
        start = std::clock();
        for (x=0; x< 1000; x++)  {
            alenkaExecute("A1 := SELECT  count(n_name) AS col1 FROM nation;\n DISPLAY A1 USING ('|');");
        }
        cout<< "Ave QPS is : " <<  ( 1000/ (( std::clock() - start ) / (double)CLOCKS_PER_SEC )) << endl;
        alenkaClose();
    }
    else {				// ordinary alenka file mode
        cout << "Executing file:" << endl;
	return execute_file( ac, av) ;
    }
}


