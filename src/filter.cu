#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "filter.h"
#include "zone_map.h"
#include <iomanip>
#include <iostream> 
#include <sstream>  

struct cmp_functor_dict
{
    const unsigned long long* source;
    bool *dest;
    const unsigned int *pars;

    cmp_functor_dict(const unsigned long long int* _source, bool * _dest,  const unsigned int * _pars):
        source(_source), dest(_dest), pars(_pars) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int idx = pars[0];
        unsigned int cmp = pars[1];
        unsigned int bits = ((unsigned int*)source)[1];
        unsigned int fit_count = ((unsigned int*)source)[0];
        unsigned int int_sz = 64;

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = ((fit_count-src_loc)-1)*bits;
        unsigned long long int tmp = source[src_idx+2]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);
        //printf("COMP1 %llu %d \n", tmp, idx);
        if(cmp == 4) { // ==
            if(tmp == idx)
                dest[i] = 1;
            else
                dest[i] = 0;
        }
        else  { // !=
            if(tmp == idx)
                dest[i] = 0;
            else
                dest[i] = 1;
        };
    }
};


struct gpu_regex
{
    char  *source;
    char *pattern;
    bool * dest;
    const unsigned int *len;

    gpu_regex(char * _source,char * _pattern, bool * _dest,
              const unsigned int * _len):
        source(_source), pattern(_pattern), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        bool star = 0;
        int j = 0;
        char* s;
        char* p;
        char* str = source + len[0]*i;
        char* pat = pattern;

loopStart:
        for (s = str, p = pat; j < len[0] && *s; ++s, ++p, ++j) {
            switch (*p) {
            case '?':
                if (*s == '.') goto starCheck;
                break;
            case '%':
                star = 1;
                str = s, pat = p;
                do {
                    ++pat;
                }
                while (*pat == '%');
                if (!*pat) {
                    dest[i] = 1;
                    return;
                }
                goto loopStart;
            default:
                if (*s != *p)
                    goto starCheck;
                break;
            } /* endswitch */
        } /* endfor */
        while (*p == '%') ++p;
        dest[i] = !*p;
        return;

starCheck:
        if (!star) {
            dest[i] = 0;
            return;
        };
        str++;
        j++;
        goto loopStart;
    }
};



bool* filter(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a,
             unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<float_type*> exe_vectors_f;
	stack<unsigned int> exe_precision;
    stack<int_type> exe_nums;
    stack<bool*> bool_vectors;
    stack<float_type> exe_nums_f;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    float_type n1_f, n2_f, res_f;
	
    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
        //cout << endl << ss << " " <<  op_nums.size() << " " << op_nums_precision.size() << endl;

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0 || ss.compare("FIELD") == 0) {


            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
                exe_type.push(ss);
				exe_precision.push(op_nums_precision.front());
				op_nums_precision.pop();
            }
            else if (ss.compare("NAME") == 0 || ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
                exe_type.push(ss);
            }
            else if(ss.compare("FIELD") == 0) {
                size_t pos1 = op_value.front().find_first_of(".", 0);
                string tbl = op_value.front().substr(0,pos1);
                string field = op_value.front().substr(pos1+1, string::npos);
                op_value.pop();
                CudaSet *b = varNames.find(tbl)->second;
                if(b->type[field] == 0) {
                    auto val = b->h_columns_int[field][0];
                    exe_nums.push(val);
                    exe_type.push("NUMBER");
                }
                if(b->type[field] == 1) {
                    auto val = b->h_columns_float[field][0];
                    exe_nums_f.push(val);
                    exe_type.push("FLOAT");
                }
                else { // not for now

                };
            }
            else if (ss.compare("FLOAT") == 0) {
                exe_nums_f.push(op_nums_f.front());
                op_nums_f.pop();
                exe_type.push(ss);
            }

        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();				
				
								
				if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_int_by_name(s1_val);
										
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,1,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,1,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));					
								exe_vectors.push(a->op(t,val*60*1000,ss,1,0,0));						
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,1,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}	
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,1,0,0));						
											}
										}		
										
									};									
								};
							};

						};
					}
					
					exe_type.push("VECTOR");
					exe_precision.push(0);

				}
				else if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_int_by_name(s1_val);
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,0,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,0,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,0,0,0));						
							}		
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,0,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}		
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,0,0,0));						
											}											
										}		
									};									
								};
							};
							
						};
					}
					exe_type.push("VECTOR");
					exe_precision.push(0);				
				}
				
				else if (s1.compare("STRING") == 0 && s2.compare("STRING") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val, tt;
					bool reverse = 1;
					
					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						tt = curr_time;								
					}
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
						}
						else {
							reverse = 0;
							pos = s1_val.find("date()");
							if(pos != string::npos) {
								tt = curr_time;								
							}
							else {
								pos = s1_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s1_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s1_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s1_val.substr(8,2));
									tm.tm_hour = std::stoi(s1_val.substr(11,2));
									tm.tm_min = std::stoi(s1_val.substr(14,2));
									tm.tm_sec = std::stoi(s1_val.substr(17,2));								
									#ifdef _WIN64
									tt = _mkgmtime (&tm);
									#else
									tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s1_val.substr(20,3));					
								}
							};							
						}	
					};


					
					
					pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = (add_interval(tt/1000, 0, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0) - tt/1000)*1000;
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = (add_interval(tt/1000, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0, 0) - tt/1000)*1000;
										}		
										else {
											pos = s1_val.find("DAY");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
											}					
											else {
												pos = s1_val.find("HOUR");
												if(pos != string::npos) {
													val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
												}											
												else {
													pos = s1_val.find("MINUTE");
													if(pos != string::npos) {
														val = stoi(s1_val.substr(0, pos)) * 60*1000;								
													}											
													else {
														pos = s1_val.find("MSECOND");
														if(pos != string::npos) {
															val = stoi(s1_val.substr(0, pos));
														}											
														else {
															pos = s1_val.find("MONTH");
															if(pos != string::npos) {
																val = stoi(s1_val.substr(0, pos));
																val = (add_interval(tt/1000, 0, val, 0, 0, 0, 0) - tt/1000)*1000;
															}						
															else {
																pos = s1_val.find("YEAR");
																if(pos != string::npos) {
																	val = stoi(s1_val.substr(0, pos));
																	val = (add_interval(tt/1000, val, 0, 0, 0, 0, 0)- tt/1000)*1000;
																}	
																else {
																	pos = s2_val.find("SECOND");
																	if(pos != string::npos) {
																		val = stoi(s2_val.substr(0, pos))*1000;
																	}	
																	else {
																		pos = s1_val.find("SECOND");
																		if(pos != string::npos) {
																			val = stoi(s1_val.substr(0, pos))*1000;
																		}	
																	}		
																}			
															};									
														};
													};							
												};
											}
										}	
									};									
								};
							};
						}	
					};				

				
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+tt;
                    else {
						if(!reverse)
							res = val-tt;
						else
							res = tt- val;
					};	

                    exe_type.push("NUMBER");
                    exe_nums.push(res);		
					exe_precision.push(0);					
				}				
				
				else if (s1.compare("STRING") == 0 && s2.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = val-n1;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				else if (s2.compare("STRING") == 0 && s1.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = n1-val;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_int_by_name(s1_val);
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,1,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,1,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,1,0,0));						
							}				
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,1,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}							
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,1,0,0));						
											}											
										}
									};									
								};
							};
	
						};
					}
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				else if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = a->get_int_by_name(s2_val);
					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,0,0,0));
					}
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,0,0,0));						
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,0,0,0));						
							}	
							else {
								pos = s1_val.find("SECOND");
								if(pos != string::npos) {
									val = stoi(s1_val.substr(0, pos));
									exe_vectors.push(a->op(t,val*1000,ss,0,0,0));						
								}											
								else {
									pos = s1_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s1_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s1_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s1_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}							
										else {
											pos = s1_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos));
												exe_vectors.push(a->op(t,val,ss,0,0,0));						
											}											
										}
									};									
								};
							};							
						};						
					}
					
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}				
				
				

                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
					
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);
					if(p1) 
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,p2);

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;

                    thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                    exe_type.push("VECTOR");
                    exe_vectors.push(thrust::raw_pointer_cast(p));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (ss.compare("ADD") == 0 )
                        res_f = n1_f+n2_f;
                    else if (ss.compare("MUL") == 0 )
                        res_f = n1_f*n2_f;
                    else if (ss.compare("DIV") == 0 )
                        res_f = n1_f/n2_f;
                    else
                        res_f = n1_f-n2_f;

                    thrust::device_ptr<float_type> p = thrust::device_malloc<float_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res_f,(float_type)0);

                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(thrust::raw_pointer_cast(p));

                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    };

                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = a->decimal_zeroes[s1_val];					

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,1));

                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,1, p1, p2));
                    };
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = a->decimal_zeroes[s2_val];					


                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
						auto pres = precision_func(p2, p1, ss);	
						exe_precision.push(pres);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,0, p2, p1));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t1 = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);
							auto p1 = a->decimal_zeroes[s1_val];					
							auto p2 = a->decimal_zeroes[s2_val];												
							auto pres = precision_func(p1, p2, ss);	
							exe_precision.push(pres);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,t1,ss,0,p2,p1));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        };
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,t1,ss,0));
                        };
                    }
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto p1 = exe_precision.top();
							exe_precision.pop();
							auto p2 = a->decimal_zeroes[s2_val];					
							auto pres = precision_func(p1, p2, ss);	
							exe_precision.push(pres);
                            exe_vectors.push(a->op(t,s3,ss,0,p1,p2));
                            hipFree(s3);

                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
							auto p1 = exe_precision.top();
							exe_precision.pop();
							auto p2 = a->decimal_zeroes[s1_val];					
							auto pres = precision_func(p2, p1, ss);	
							exe_precision.push(pres);
                            exe_vectors.push(a->op(t,s3,ss,1,p2,p1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);						
                        exe_vectors.push(a->op(s3,n1, ss,1,p1,p2));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") || s2.compare("VECTOR F") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto pres = precision_func(p2, p1, ss);	
						exe_precision.push(pres);												
                        exe_vectors.push(a->op(s3,n1, ss,0, p2, p1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,0));
                        hipFree(s3);
                    }
                }
				
				else if (s1.compare("VECTOR") == 0 && s2.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    int_type* s4 = exe_vectors.top();
					exe_vectors.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(s4,val*24*60*60*1000,ss,1,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(s4,val*60*60*1000,ss,1,0,0));
						}
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(s4,val*60*1000,ss,1,0,0));
							}
							else {
								pos = s2_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;							
									exe_vectors.push(a->op(s4,val,ss,1,0,0));									
								}	
								else {
									pos = s1_val.find("MSECOND");
									if(pos != string::npos) {
										val = stoi(s1_val.substr(0, pos));
										exe_vectors.push(a->op(s4,val,ss,1,0,0));						
									}											
									else {
										pos = s1_val.find("MONTH");
										if(pos != string::npos) {
											val = stoi(s1_val.substr(0, pos));
											if (ss.compare("ADD") != 0 )
												val = -val;
											thrust::device_ptr<int_type> dev_ptr(s4);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(s4);																
										}						
										else {
											pos = s1_val.find("YEAR");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos));
												int_type* temp = (int_type*)malloc(2*int_size);
												if (ss.compare("ADD") != 0 )
													val = -val;											
												thrust::device_ptr<int_type> dev_ptr(s4);
												thrust::host_vector<int_type> tt(a->mRecCount);
												thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
												
												for(int z = 0; z < a->mRecCount; z++) {
													tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
												};	
												thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
												exe_vectors.push(s4);																
											}							
											else {
												pos = s1_val.find("SECOND");
												if(pos != string::npos) {
													val = stoi(s1_val.substr(0, pos));
													exe_vectors.push(a->op(s4,val*1000,ss,1,0,0));						
												}											
											}	
										}			
									};
								};
								
							}																		

						};
					}
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				
				else if (s2.compare("VECTOR") == 0 && s1.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* s4 = exe_vectors.top();
					exe_vectors.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(s4,val*24*60*60*1000,ss,0,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(s4,val*60*60*1000,ss,0,0,0));
						}
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(s4,val*60*1000,ss,0,0,0));
							}
							else {
								pos = s2_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;							
									exe_vectors.push(a->op(s4,val,ss,0,0,0));									
								}	
								else {
									pos = s2_val.find("MSECOND");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										exe_vectors.push(a->op(s4,val,ss,1,0,0));						
									}											
									else {
										pos = s2_val.find("MONTH");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											if (ss.compare("ADD") != 0 )
												val = -val;
											thrust::device_ptr<int_type> dev_ptr(s4);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(s4);																
										}						
										else {
											pos = s2_val.find("YEAR");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												int_type* temp = (int_type*)malloc(2*int_size);
												if (ss.compare("ADD") != 0 )
													val = -val;											
												thrust::device_ptr<int_type> dev_ptr(s4);
												thrust::host_vector<int_type> tt(a->mRecCount);
												thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
												
												for(int z = 0; z < a->mRecCount; z++) {
													tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
												};	
												thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
												exe_vectors.push(s4);																
											}							
											else {
												pos = s2_val.find("SECOND");
												if(pos != string::npos) {
													val = stoi(s2_val.substr(0, pos));
													exe_vectors.push(a->op(s4,val*1000,ss,1,0,0));						
												}											
											}
										}									
									};
								};
								
							}
						};
					}

					
					exe_type.push("VECTOR");
					exe_precision.push(0);
				}
				

                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s4 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);						
					
                    exe_vectors.push(a->op(s3, s4,ss,1,p1,p2));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,0));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
            }

            else if (ss.compare("CMP") == 0) {


                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();

                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);				
					if(p1) 
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,p2);
					
                    bool_vectors.push(a->compare(n1,n2,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,n2_f,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NUMBER") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }

                else if ((s1.compare("STRING") == 0 && s2.compare("NAME") == 0) ||
                         (s1.compare("NAME") == 0 && s2.compare("STRING") == 0))
                {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					
					if(s1_val == "date()" || s2_val == "date()") {
						if(s1.compare("STRING") == 0) {
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = a->get_int_by_name(s1_val);
							
							auto pos = s2_val.find("date()");
							if(pos != string::npos) {
								val = curr_time;
								bool_vectors.push(a->compare(t,val,cmp_type,0,0));
							}		
							else {
								pos = s2_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s2_val.substr(8,2));
									tm.tm_hour = std::stoi(s2_val.substr(11,2));
									tm.tm_min = std::stoi(s2_val.substr(14,2));
									tm.tm_sec = std::stoi(s2_val.substr(17,2));								
									#ifdef _WIN64
									auto tt = _mkgmtime (&tm);
									#else
									auto tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
									bool_vectors.push(a->compare(t,tt, cmp_type,0,0));
								}
							};
							
							exe_type.push("VECTOR");
							exe_precision.push(0);												
						}
						else {
							s2_val = exe_value.top();
							exe_value.pop();
							s1_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = a->get_int_by_name(s1_val);
							
							auto pos = s2_val.find("date()");
							if(pos != string::npos) {
								val = curr_time;
								bool_vectors.push(a->compare(t,val, cmp_type,0,0));
							}					
							else {
								pos = s2_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s2_val.substr(8,2));
									tm.tm_hour = std::stoi(s2_val.substr(11,2));
									tm.tm_min = std::stoi(s2_val.substr(14,2));
									tm.tm_sec = std::stoi(s2_val.substr(17,2));								
									#ifdef _WIN64
									auto tt = _mkgmtime (&tm);
									#else
									auto tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
									bool_vectors.push(a->compare(t,tt, cmp_type,0,0));
								}
							};

							exe_type.push("VECTOR");
							exe_precision.push(0);					
						}						
					}
					else {	

						if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
							s1.swap(s2);
							s1_val.swap(s2_val);
						};
						
						if (a->type[s2_val] == 0 && a->ts_cols[s2_val] ) {
							struct std::tm tm;						
							auto year = s1_val.substr(0,4);
							auto month = s1_val.substr(5,2);
							auto day = s1_val.substr(8,2);
							auto hour = s1_val.substr(11,2);
							auto min = s1_val.substr(14,2);
							auto sec = s1_val.substr(17,2);
							auto usec = s1_val.substr(20,3);
							//cout << "VL " << year << " " << month << " " << day << " " << hour << " " << min << " " << sec << " " << usec << "   " << endl;
							
							tm.tm_year = std::stoi(year)-1900;
							tm.tm_mon = std::stoi(month)-1;
							tm.tm_mday = std::stoi(day);
							tm.tm_hour = std::stoi(hour);
							tm.tm_min = std::stoi(min);
							tm.tm_sec = std::stoi(sec);
							
							#ifdef _WIN64
							time_t time = _mkgmtime (&tm);
							#else
							time_t time = timegm (&tm);
							#endif
							
							
							time = time*1000 + std::stoi(usec);					
							int_type* t = a->get_int_by_name(s2_val);
							exe_precision.push(0);						
							exe_type.push("VECTOR");
							bool_vectors.push(a->compare(t,(int_type)time,cmp_type, 0, 0));						

						}
						else {	
						
							void* d_res, *d_v;
							if(cmp_type != 7)
								hipMalloc((void **) &d_res, a->mRecCount);
							else
								hipMalloc((void **) &d_res, a->hostRecCount);
							thrust::device_ptr<bool> dd_res((bool*)d_res);

							hipMalloc((void **) &d_v, 8);
							thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
							thrust::counting_iterator<unsigned int> begin(0);

							if(s2_val.find(".") != string::npos) { //bitmap index
								auto pos1 = s2_val.find_first_of(".");
								auto pos2 = s2_val.find_last_of(".");
								auto set = s2_val.substr(pos1+1, (pos2-pos1)-1);
								auto col = s2_val.substr(pos2+1);
								auto len = data_dict[set][col].col_length;								
								
								while(s1_val.length() < len)
									s1_val = s1_val + '\0';

								auto s1_hash = MurmurHash64A(&s1_val[0], len, hash_seed)/2;								
								if(a->idx_dictionary_int[s2_val].find(s1_hash) != a->idx_dictionary_int[s2_val].end()) {
									dd_v[0] = a->idx_dictionary_int[s2_val][s1_hash];
									dd_v[1] = (unsigned int)cmp_type;
									cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
									thrust::for_each(begin, begin + a->mRecCount, ff);
								}
								else {
									hipMemset(d_res,0,a->mRecCount);
								}
							}
							else {

								auto s = a->string_map[s2_val];
								auto pos = s.find_first_of(".");
								auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

								dd_v[0] = len;
								dd_v[1] = (unsigned int)s1_val.length() + 1;

								if(cmp_type != 7) {
									thrust::device_vector<unsigned long long int> vv(1);
									while(s1_val.length() < len) {
										s1_val = s1_val + '\0';
									};

									vv[0] = MurmurHash64A(&s1_val[0], s1_val.length(), hash_seed)/2;

									string f1 = a->load_file_name + "." + s2_val + "." + to_string(segment) + ".hash";
									FILE* f = fopen(f1.c_str(), "rb" );
									unsigned long long int* buff = new unsigned long long int[a->mRecCount];
									unsigned int cnt;
									fread(&cnt, 4, 1, f);
									fread(buff, a->mRecCount*8, 1, f);
									fclose(f);
									thrust::device_vector<unsigned long long int> vals(a->mRecCount);
									thrust::copy(buff, buff+a->mRecCount, vals.begin());
									if(cmp_type == 4) //==
										thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::equal_to<unsigned long long int>());
									else if(cmp_type == 3) //!=
										thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::not_equal_to<unsigned long long int>());
									delete [] buff;

								}
								else {
									if(a->map_like.find(s2_val) == a->map_like.end()) {
								
										void* d_str;
										hipMalloc((void **) &d_str, len);
										hipMemset(d_str,0,len);
										hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);
										
										string f1 = a->load_file_name + "." + s2_val;
										FILE* f = fopen(f1.c_str(), "rb" );
										fseek(f, 0, SEEK_END);
										long fileSize = ftell(f);
										fseek(f, 0, SEEK_SET);																
																		
										unsigned int pieces = 1;
										if(fileSize > getFreeMem()/2)
											pieces = fileSize /(getFreeMem()/2) + 1;
										auto piece_sz = fileSize/pieces;
										ldiv_t ldivresult = ldiv(fileSize/pieces, len);		
										if(ldivresult.rem != 0)
											piece_sz = fileSize/pieces + (len - ldivresult.rem);										
										thrust::device_vector<char> dev(piece_sz);	
										char* buff = new char[piece_sz];
										a->map_res[s2_val] = thrust::device_vector<unsigned int>();
										for(auto i = 0; i < pieces; i++) {	
											
											if(i == pieces-1)
												piece_sz = fileSize - piece_sz*i;											
											fread(buff, piece_sz, 1, f);	
											hipMemcpy( thrust::raw_pointer_cast(dev.data()), (void*)buff, piece_sz, hipMemcpyHostToDevice);

											gpu_regex ff(thrust::raw_pointer_cast(dev.data()), (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
											thrust::for_each(begin, begin + piece_sz/len, ff);
											
											auto cnt = thrust::count(dd_res, dd_res + piece_sz/len, 1);
											auto offset = a->map_res[s2_val].size();
											a->map_res[s2_val].resize(a->map_res[s2_val].size() + cnt);
											thrust::copy_if(thrust::make_counting_iterator((unsigned int)(i*(piece_sz/len))), thrust::make_counting_iterator((unsigned int)((i+1)*(piece_sz/len))),
															dd_res, a->map_res[s2_val].begin() + offset, thrust::identity<bool>());
										};				
										
										fclose(f);
										delete [] buff;												
										hipFree(d_str);
										thrust::sort(a->map_res[s2_val].begin(), a->map_res[s2_val].end());
										a->map_like[s2_val] = 1;


									};
									// now lets calc the current segments's matches
									hipMemset(d_res, 0, a->hostRecCount);
									binary_search(a->map_res[s2_val].begin(),a->map_res[s2_val].end(), a->d_columns_int[s2_val].begin(), a->d_columns_int[s2_val].end(), dd_res);
								};
							};

							hipFree(d_v);
							exe_type.push("VECTOR");
							bool_vectors.push((bool*)d_res);
						}	
					}
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {					
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(s1_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);
                        if(a->idx_dictionary_int[s1_val].find(n1) != a->idx_dictionary_int[s1_val].end()) {
                            dd_v[0] = a->idx_dictionary_int[s1_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;
                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(idx_vals[s1_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);							
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("VECTOR");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
                        if (a->type[s1_val] == 0) {
                            int_type* t = a->get_int_by_name(s1_val);
							thrust::device_ptr<int_type> bp((int_type*)t);
							auto p2 = exe_precision.top();
							exe_precision.pop();
							auto p1 = a->decimal_zeroes[s1_val];			
							auto pres = std::max(p1, p2);	
							exe_precision.push(pres);
							
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,n1,cmp_type, pres-p1, pres-p2));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                        };
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if(s2_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);

                        if(a->idx_dictionary_int[s2_val].find(n1) != a->idx_dictionary_int[s2_val].end()) {

                            dd_v[0] = a->idx_dictionary_int[s2_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;

                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("VECTOR");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);
							auto p2 = exe_precision.top();
							exe_precision.pop();
							auto p1 = a->decimal_zeroes[s2_val];					
							auto pres = std::max(p1, p2);	
							exe_precision.push(pres);							
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,n1,cmp_type, p1, p2));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR");
                            bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                        };
                    };
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type,0,0));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type,0,0));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
					
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = std::max(p1, p2);	
					exe_precision.push(pres);

                    bool_vectors.push(a->compare(s3,n1,cmp_type, p1, p2));
                    hipFree(s3);
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR F") == 0) {

                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }
				
				else if (s1.compare("VECTOR") == 0 && s2.compare("STRING") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					exe_precision.pop();
                    bool_vectors.push(a->compare(s3,curr_time,cmp_type,0,0));
                    hipFree(s3);
					exe_precision.push(0);					
				}
				
				else if (s2.compare("VECTOR") == 0 && s1.compare("STRING") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");
					exe_precision.pop();
                    bool_vectors.push(a->compare(s3,curr_time,cmp_type,0,0));
                    hipFree(s3);
					exe_precision.push(0);						
				}
				

                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") == 0) {

                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = std::max(p1, p2);	
					exe_precision.push(pres);
					
                    bool_vectors.push(a->compare(s3,n1,cmp_type, p2, p1));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("VECTOR") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type,0,0));
                    hipFree(s3);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type,0,0));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NAME") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }


                else if (s1.compare("VECTOR") == 0 && s2.compare("NAME") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
						
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = a->decimal_zeroes[s2_val];					
						auto pres = max(p1, p2);	
                        bool_vectors.push(a->compare(t,s3,cmp_type,p2,p1));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR F") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
						auto p1 = exe_precision.top();
						exe_precision.pop();
						auto p2 = a->decimal_zeroes[s2_val];				
                        bool_vectors.push(a->compare(t,s3,cmp_type,p1,p2));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();
					auto pres = max(p1, p2);	
					exe_precision.push(pres);
					
                    bool_vectors.push(a->compare(s2,s3,cmp_type, p2, p1));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s2 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }


                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s1_val] == 0 || a->type[s1_val] == 2) {
                        int_type* t = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
							auto p1 = a->decimal_zeroes[s1_val];					
							auto p2 = a->decimal_zeroes[s2_val];					
							auto pres = max(p1, p2);	
							exe_precision.push(pres);
                            bool_vectors.push(a->compare(t1,t,cmp_type, p2, p1));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t1,t,cmp_type));
                        };
                    }
                    else {
                        cmp_type = reverse_op(cmp_type);
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        };
                    }
                }
            }

            else if (ss.compare("AND") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_or(s2,s3));
            }
            else {
                cout << "found nothing " << endl;
            }
        };
    };

    return bool_vectors.top();
}